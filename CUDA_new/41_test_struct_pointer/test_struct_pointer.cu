#include <iostream>
#include <hip/hip_runtime.h>

struct MyStruct {
    int* data;
    int size;
};

__global__ void deviceKernel(MyStruct* deviceStruct) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // 在设备上访问设备上分配的内存
    if (tid < deviceStruct->size) {
        printf("Device access: %d\n", deviceStruct->data[tid]);
    }
}

int main() {
    const int size = 5;

    // 在主机上分配内存
    int* hostData = new int[size];
    for (int i = 0; i < size; ++i) {
        hostData[i] = i;
    }

    // 创建结构体并设置指针
    MyStruct hostStruct;
    hostStruct.size = size;

    // 在设备上分配内存以存储结构体的数据
    hipMalloc((void**)&hostStruct.data, size * sizeof(int));

    // 将数据从主机拷贝到设备
    hipMemcpy(hostStruct.data, hostData, size * sizeof(int), hipMemcpyHostToDevice);

    // 在设备上分配内存以存储结构体
    MyStruct* deviceStruct;
    hipMalloc((void**)&deviceStruct, sizeof(MyStruct));

    // 将结构体从主机拷贝到设备
    hipMemcpy(deviceStruct, &hostStruct, sizeof(MyStruct), hipMemcpyHostToDevice);

    // 在设备上执行核函数
    deviceKernel<<<1, size>>>(deviceStruct);

    // 等待设备执行完成
    hipDeviceSynchronize();

    // 释放内存
    delete[] hostData;
    hipFree(hostStruct.data);
    hipFree(deviceStruct);

    return 0;
}

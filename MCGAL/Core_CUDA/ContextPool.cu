#include "core.h"

namespace MCGAL {

ContextPool::ContextPool() {
    // mallocOnUnifiedMemory();
    mallocOnCpu();
}

ContextPool::~ContextPool() {
    // freeOnUnifiedMemory();
    freeOnCpu();
}

void ContextPool::mallocOnUnifiedMemory() {
    vpool = new MCGAL::Vertex*[VERTEX_POOL_SIZE];
    CHECK(hipMallocManaged(vpool, VERTEX_POOL_SIZE * sizeof(Vertex*)));
    hpool = new MCGAL::Halfedge*[HALFEDGE_POOL_SIZE];
    CHECK(hipMallocManaged(hpool, HALFEDGE_POOL_SIZE * sizeof(Halfedge*)));
    fpool = new MCGAL::Facet*[FACET_POOL_SIZE];
    CHECK(hipMallocManaged(fpool, FACET_POOL_SIZE * sizeof(Facet*)));
    for (int i = 0; i < VERTEX_POOL_SIZE; i++) {
        vpool[i] = new MCGAL::Vertex();
        CHECK(hipMallocManaged(&vpool[i], sizeof(Vertex)));
    }
    for (int i = 0; i < HALFEDGE_POOL_SIZE; i++) {
        hpool[i] = new MCGAL::Halfedge();
        CHECK(hipMallocManaged(&hpool[i], sizeof(Halfedge)));
    }
    for (int i = 0; i < FACET_POOL_SIZE; i++) {
        fpool[i] = new MCGAL::Facet();
        CHECK(hipMallocManaged(&fpool[i], sizeof(Facet)));
    }
}

void ContextPool::freeOnUnifiedMemory() {
    if (vpool != nullptr) {
        for (int i = 0; i < VERTEX_POOL_SIZE; i++) {
            hipFree(vpool[i]);
            delete vpool[i];
            vpool[i] = nullptr;
        }
        delete[] vpool;
        hipFree(vpool);
        vpool = nullptr;
    }
    if (hpool != nullptr) {
        for (int i = 0; i < HALFEDGE_POOL_SIZE; i++) {
            hipFree(hpool[i]);
            delete hpool[i];
            hpool[i] = nullptr;
        }
        hipFree(hpool);
        delete[] hpool;
        hpool = nullptr;
    }
    if (fpool != nullptr) {
        for (int i = 0; i < FACET_POOL_SIZE; i++) {
            hipFree(fpool[i]);
            delete fpool[i];
            fpool[i] = nullptr;
        }
        hipFree(fpool);
        delete[] fpool;
        fpool = nullptr;
    }
}

void ContextPool::mallocOnCpu() {
    vpool = new MCGAL::Vertex*[VERTEX_POOL_SIZE];
    hpool = new MCGAL::Halfedge*[HALFEDGE_POOL_SIZE];
    fpool = new MCGAL::Facet*[FACET_POOL_SIZE];
    for (int i = 0; i < VERTEX_POOL_SIZE; i++) {
        vpool[i] = new MCGAL::Vertex();
    }
    for (int i = 0; i < HALFEDGE_POOL_SIZE; i++) {
        hpool[i] = new MCGAL::Halfedge();
    }
    for (int i = 0; i < FACET_POOL_SIZE; i++) {
        fpool[i] = new MCGAL::Facet();
    }
}

void ContextPool::freeOnCpu() {
    if (vpool != nullptr) {
        for (int i = 0; i < VERTEX_POOL_SIZE; i++) {
            delete vpool[i];
            vpool[i] = nullptr;
        }
        delete[] vpool;
        vpool = nullptr;
    }
    if (hpool != nullptr) {
        for (int i = 0; i < HALFEDGE_POOL_SIZE; i++) {
            delete hpool[i];
            hpool[i] = nullptr;
        }
        delete[] hpool;
        hpool = nullptr;
    }
    if (fpool != nullptr) {
        for (int i = 0; i < FACET_POOL_SIZE; i++) {
            delete fpool[i];
            fpool[i] = nullptr;
        }
        delete[] fpool;
        fpool = nullptr;
    }
}

void ContextPool::copyToCuda() {
    hipMalloc((void**)&dvpool, VERTEX_POOL_SIZE * sizeof(Vertex*));
    for (int i = 0; i < VERTEX_POOL_SIZE; ++i) {
        hipMalloc((void**)&(dvpool[i]), sizeof(Vertex));
        hipMemcpy(dvpool[i], vpool[i], sizeof(Vertex), hipMemcpyHostToDevice);
    }

    hipMalloc((void**)&dhpool, HALFEDGE_POOL_SIZE * sizeof(Halfedge*));
    for (int i = 0; i < HALFEDGE_POOL_SIZE; ++i) {
        hipMalloc((void**)&(dhpool[i]), sizeof(Halfedge));
        hipMemcpy(dhpool[i], hpool[i], sizeof(Halfedge), hipMemcpyHostToDevice);
    }

    hipMalloc((void**)&dfpool, FACET_POOL_SIZE * sizeof(Facet*));
    for (int i = 0; i < FACET_POOL_SIZE; ++i) {
        hipMalloc((void**)&(dfpool[i]), sizeof(Facet));
        hipMemcpy(dfpool[i], fpool[i], sizeof(Facet), hipMemcpyHostToDevice);
    }
}

void ContextPool::freeCuda() {
    for (int i = 0; i < VERTEX_POOL_SIZE; ++i) {
        hipFree(dvpool[i]);
    }
    hipFree(dvpool);

    for (int i = 0; i < HALFEDGE_POOL_SIZE; ++i) {
        hipFree(dhpool[i]);
    }
    hipFree(dhpool);

    for (int i = 0; i < FACET_POOL_SIZE; ++i) {
        hipFree(dfpool[i]);
    }
    hipFree(dfpool);
}

}  // namespace MCGAL
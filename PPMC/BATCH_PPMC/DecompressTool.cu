#include "hip/hip_runtime.h"
#include "DecompressTool.cuh"
#include <thread>
__global__ void
readBaseMeshOnCuda(char* buffer, int* stOffsets, int num, int* vh_departureConquest, int* nbDecimations) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < num) {}
}

__global__ void warmup() {
    int tid1 = 1;
    int tid2 = 2;
    int tid3 = tid1 + tid2;
}

DeCompressTool::~DeCompressTool() {
    stOffsets.clear();
    stOffsets.shrink_to_fit();
    lods.clear();
    lods.shrink_to_fit();
    nbDecimations.clear();
    nbDecimations.shrink_to_fit();
    vh_departureConquest.clear();
    vh_departureConquest.shrink_to_fit();
    splitableCounts.clear();
    splitableCounts.shrink_to_fit();
    insertedCounts.clear();
    insertedCounts.shrink_to_fit();

    delete[] buffer;
    hipFree(dbuffer);
    hipFree(dvh_departureConquest);
    hipFree(dstOffsets);
    hipFree(dfaceIndexes);
    hipFree(dvertexIndexes);
    hipFree(dstHalfedgeIndexes);
    hipFree(dstFacetIndexes);
}

/**
 * 思考一种比较好的处理方式，是从多个路径读取多个文件还是从一个读出来然后解析
 * 这里确定一下，从多个文件里读取
 */
DeCompressTool::DeCompressTool(char** path, int number, bool is_base) {
    int dataOffset = 0;
    buffer = new char[BUFFER_SIZE];
    for (int i = 0; i < number; i++) {
        std::ifstream fin(path[i], std::ios::binary);
        int len2;
        fin.read((char*)&len2, sizeof(int));
        char* p_data = new char[len2];
        memset(p_data, 0, len2);
        stOffsets.push_back(dataOffset);
        fin.read(p_data, len2);
        memcpy(buffer + dataOffset, p_data, len2);
        dataOffset += len2;
        free(p_data);
    }
    CHECK(hipMalloc((int**)&dfaceIndexes, SPLITABLE_SIZE * sizeof(int)));
    CHECK(hipMalloc((int**)&dvertexIndexes, SPLITABLE_SIZE * sizeof(int)));
    CHECK(hipMalloc((int**)&dstHalfedgeIndexes, SPLITABLE_SIZE * sizeof(int)));
    CHECK(hipMalloc((int**)&dstFacetIndexes, SPLITABLE_SIZE * sizeof(int)));
    CHECK(hipMalloc(&dbuffer, dataOffset));
    CHECK(hipMalloc(&dstOffsets, stOffsets.size() * sizeof(int)));
    CHECK(hipMalloc(&dSplittabelCount, stOffsets.size() * sizeof(int)));
    CHECK(hipMemcpy(dbuffer, buffer, dataOffset, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dstOffsets, stOffsets.data(), stOffsets.size() * sizeof(int), hipMemcpyHostToDevice));
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("ERROR: %s:%d,", __FILE__, __LINE__);
        printf("code:%d,reason:%s\n", error, hipGetErrorString(error));
        exit(1);
    }
    batch_size = number;
    if (is_base) {
        vh_departureConquest.resize(2 * number);
        nbDecimations.resize(number);
        splitableCounts.resize(number);
        insertedCounts.resize(number);
        dim3 block(256, 1, 1);
        dim3 grid((number + block.x - 1) / block.x, 1, 1);
#pragma omp parallel for
        for (int i = 0; i < number; i++) {
            readBaseMesh(i, &stOffsets[i]);
            if (stOffsets[i] % 4 != 0) {
                stOffsets[i] = (stOffsets[i] / 4 + 1) * 4;
            }
        }
        hipMemcpy(dstOffsets, stOffsets.data(), batch_size * sizeof(int), hipMemcpyHostToDevice);
    }
}

void DeCompressTool::decode(int lod) {
    if (lod < i_decompPercentage) {
        return;
    }
    i_decompPercentage = lod;
    b_jobCompleted = false;
    warmup<<<16, 256>>>();
    hipDeviceSynchronize();
    while (!b_jobCompleted) {
        startNextDecompresssionOp();
    }
}

__global__ void resetStateOnCuda(MCGAL::Halfedge* hpool, MCGAL::Facet* fpool, int num) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < num) {
        MCGAL::Facet* fit = &fpool[tid];
        if (fit->isRemovedOnCuda()) {
            fit->setMeshIdOnCuda(-1);
            for (int i = 0; i < fit->halfedge_size; i++) {
                MCGAL::Halfedge* hit = fit->getHalfedgeByIndexOnCuda(hpool, i);
                if (hit->isRemovedOnCuda()) {
                    hit->setMeshIdOnCuda(-1);
                }
                hit->resetStateOnCuda();
            }
        } else {
            fit->resetStateOnCuda();
            for (int i = 0; i < fit->halfedge_size; i++) {
                MCGAL::Halfedge* hit = fit->getHalfedgeByIndexOnCuda(hpool, i);
                if (hit->isRemovedOnCuda()) {
                    hit->setMeshIdOnCuda(-1);
                }
                hit->resetStateOnCuda();
            }
        }
    }
}

void DeCompressTool::startNextDecompresssionOp() {
    // check if the target LOD is reached
    if (i_curDecimationId * 100.0 / nbDecimations[0] >= i_decompPercentage) {
        if (i_curDecimationId == nbDecimations[0]) {}
        b_jobCompleted = true;
        return;
    }
    std::vector<int> twos;
    // 1. reset the states. note that the states of the vertices need not to be reset
    int number = *MCGAL::contextPool.findex;
    dim3 block(256, 1, 1);
    dim3 grid((number + block.x - 1) / block.x, 1, 1);
    int vsize = *MCGAL::contextPool.vindex;
    int hsize = *MCGAL::contextPool.hindex;
    int fsize = *MCGAL::contextPool.findex;
    CHECK(hipMemcpy(MCGAL::contextPool.dvpool, MCGAL::contextPool.vpool, vsize * sizeof(MCGAL::Vertex),
                     hipMemcpyHostToDevice));
    CHECK(hipMemcpy(MCGAL::contextPool.dhpool, MCGAL::contextPool.hpool, hsize * sizeof(MCGAL::Halfedge),
                     hipMemcpyHostToDevice));
    CHECK(hipMemcpy(MCGAL::contextPool.dfpool, MCGAL::contextPool.fpool, fsize * sizeof(MCGAL::Facet),
                     hipMemcpyHostToDevice));
    resetStateOnCuda<<<grid, block>>>(MCGAL::contextPool.dhpool, MCGAL::contextPool.dfpool, number);
    hipDeviceSynchronize();
    for (int i = 0; i < splitableCounts.size(); i++) {
        splitableCounts[i] = 0;
        insertedCounts[i] = 0;
    }
    i_curDecimationId++;  // increment the current decimation operation id.
    // 2. decoding the removed vertices and add to target facets
    struct timeval start = get_cur_time();
    BatchRemovedVerticesDecodingStep();
    // logt("%d RemovedVerticesDecodingStep", start, i_curDecimationId);
    // 3. decoding the inserted edge and marking the ones added
    BatchInsertedEdgeDecodingStepOnCuda();
    // std::thread thread1([&]() -> void { BatchRemovedVerticesDecodingStep(); });
    // std::thread thread2([&]() -> void { BatchInsertedEdgeDecodingStepOnCuda(); });
    // std::thread thread1(&DeCompressTool::BatchRemovedVerticesDecodingStep, this);
    // std::thread thread2(&DeCompressTool::BatchInsertedEdgeDecodingStepOnCuda, this);
    // thread1.join();
    // thread2.join();
    logt("%d InsertedEdgeDecodingStep", start, i_curDecimationId);
    // 4. truly insert the removed vertices
    insertRemovedVerticesOnCuda();
    // insertRemovedVertices();
    logt("%d insertRemovedVertices", start, i_curDecimationId);
    // 5. truly remove the added edges
#pragma omp parallel for num_threads(batch_size)
    for (int i = 0; i < batch_size; i++) {
        removeInsertedEdges(i);
    }
    logt("%d removeInsertedEdges", start, i_curDecimationId);
}

MCGAL::Halfedge* DeCompressTool::pushHehInit(int meshId) {
    MCGAL::Halfedge* hehBegin;
    MCGAL::Vertex* v1 = MCGAL::contextPool.getVertexByIndex(vh_departureConquest[meshId * 2 + 1]);
    MCGAL::Vertex* v0 = MCGAL::contextPool.getVertexByIndex(vh_departureConquest[meshId * 2]);
    for (int i = 0; i < v1->halfedges_size; i++) {
        MCGAL::Halfedge* hit = v1->getHalfedgeByIndex(i);
        if (hit->opposite()->vertex_ == vh_departureConquest[meshId * 2]) {
            hehBegin = hit->opposite();
            break;
        }
    }
    // assert(hehBegin->vertex() == vh_departureConquest[0]);
    // Push it to the queue.
    return hehBegin;
}

bool cmpForder(int fid1, int fid2) {
    MCGAL::Facet* f1 = MCGAL::contextPool.getFacetByIndex(fid1);
    MCGAL::Facet* f2 = MCGAL::contextPool.getFacetByIndex(fid2);
    if (f1->forder == ~(unsigned long long)0) {
        return false;
    } else if (f2->forder == ~(unsigned long long)0) {
        return true;
    }
    if (f1->meshId == f2->meshId) {
        return f1->forder < f2->forder;
    }
    return f1->meshId < f2->meshId;
}

bool cmpHorder(int hid1, int hid2) {
    MCGAL::Halfedge* h1 = MCGAL::contextPool.getHalfedgeByIndex(hid1);
    MCGAL::Halfedge* h2 = MCGAL::contextPool.getHalfedgeByIndex(hid2);
    if (h1->horder == ~(unsigned long long)0) {
        return false;
    } else if (h2->horder == ~(unsigned long long)0) {
        return true;
    }
    if (h1->meshId == h2->meshId) {
        return h1->horder < h2->horder;
    }
    return h1->meshId < h2->meshId;
}

void DeCompressTool::RemovedVerticesDecodingOnCuda() {
    int size = *MCGAL::contextPool.findex;
    int* fids = new int[size];
    int index = 0;
    int* fsizes = new int[batch_size];
    int* fsizesSum = new int[batch_size + 1];
    memset(fsizes, 0, batch_size * sizeof(int));
    memset(fsizesSum, 0, (batch_size + 1) * sizeof(int));
    for (int i = 0; i < size; i++) {
        if (MCGAL::contextPool.fpool[i].meshId != -1) {
            fids[index++] = i;
            fsizes[MCGAL::contextPool.fpool[i].meshId]++;
        }
    }
    for (int i = 1; i < batch_size; i++) {
        fsizesSum[i] = fsizesSum[i - 1] + fsizes[i];
    }
    fsizesSum[batch_size] = index;
    int* firstQueue = new int[size];
    int* secondQueue = new int[size];
    int currentQueueSize = batch_size;
    int nextQueueSize = 0;
    int level = 0;
    for (int i = 0; i < batch_size; i++) {
        MCGAL::Halfedge* hit = pushHehInit(i);
        hit->facet()->forder = 0;
        firstQueue[i] = hit->poolId;
    }
}

__global__ void initForder(MCGAL::Facet* fpool, int* ids, int num) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < num) {
        int fid = ids[tid];
        fpool[fid].forder = 0;
    }
}

__global__ void initHorder(MCGAL::Halfedge* hpool, int* ids, int num) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < num) {
        int hid = ids[tid];
        hpool[hid].horder = 0;
    }
}

void DeCompressTool::InsertedEdgeDecodingOnCuda() {}

__global__ void computeFacetNextQueue(MCGAL::Vertex* vpool,
                                      MCGAL::Halfedge* hpool,
                                      MCGAL::Facet* fpool,
                                      int* currentQueue,
                                      int* nextQueue,
                                      int* nextQueueSize,
                                      int currentQueueSize) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < currentQueueSize) {
        int current = currentQueue[tid];
        MCGAL::Halfedge* h = &hpool[current];
        MCGAL::Facet* f = h->dfacet(fpool);
        if (f->isProcessedOnCuda()) {
            return;
        }
        MCGAL::Halfedge* hIt = h;
        unsigned long long idx = 1;
        do {
            MCGAL::Halfedge* hOpp = hIt->dopposite(hpool);
            unsigned long long order = f->forder << 4 | idx;
            atomicMin(&hOpp->dfacet(fpool)->forder, order);
            if (hOpp->dfacet(fpool)->forder == order && !hOpp->dfacet(fpool)->isProcessedOnCuda()) {
                idx++;
                if (hOpp->dfacet(fpool)->indexInQueue != -1) {
                    nextQueue[hOpp->dfacet(fpool)->indexInQueue] = hOpp->poolId;
                } else {
                    int position = atomicAdd(nextQueueSize, 1);
                    hOpp->dfacet(fpool)->indexInQueue = position;
                    nextQueue[position] = hOpp->poolId;
                }
            }
            hIt = hIt->dnext(hpool);
        } while (hIt != h);
    }
}

__global__ void computeHalfedgeNextQueue(MCGAL::Vertex* vpool,
                                         MCGAL::Halfedge* hpool,
                                         MCGAL::Facet* fpool,
                                         int* currentQueue,
                                         int* nextQueue,
                                         int* nextQueueSize,
                                         int currentQueueSize) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < currentQueueSize) {
        int current = currentQueue[tid];
        MCGAL::Halfedge* h = &hpool[current];
        if (h->isProcessedOnCuda()) {
            return;
        }
        MCGAL::Halfedge* hIt = h->dnext(hpool);
        unsigned long long idx = 1;
        while (hIt->dopposite(hpool) != h) {
            unsigned long long order = h->horder << 4 | idx;

            atomicMin(&hIt->horder, order);

            if (hIt->horder == order) {
                idx++;
                int position = atomicAdd(nextQueueSize, 1);
                nextQueue[position] = hIt->poolId;
            }
            hIt = hIt->dopposite(hpool)->dnext(hpool);
        };
    }
}

__global__ void
setProcessedProcessedFlagOnCuda(MCGAL::Halfedge* hpool, MCGAL::Facet* fpool, int* currentQueue, int currentQueueSize) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < currentQueueSize) {
        MCGAL::Halfedge* h = &hpool[currentQueue[tid]];
        h->dfacet(fpool)->indexInQueue = -1;
        h->dfacet(fpool)->setProcessedFlagOnCuda();
    }
}

__global__ void meshIdCount(MCGAL::Facet* fpool, int* fsizes, int num) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < num) {
        atomicAdd(&(fsizes[fpool[tid].meshId]), 1);
        __syncthreads();
    }
}

__global__ void countFacetOccurrences(MCGAL::Facet* fpool, int* fids, int* fsizes, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < size) {
        atomicAdd(&(fsizes[fpool[fids[tid]].meshId]), 1);
        tid += blockDim.x * gridDim.x;
    }
}

__global__ void countHalfedgeOccurrences(MCGAL::Halfedge* hpool, int* hids, int* hsizes, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < size) {
        atomicAdd(&(hsizes[hpool[hids[tid]].meshId]), 1);
        tid += blockDim.x * gridDim.x;
    }
}

__global__ void readFacetSymbolOnCuda(MCGAL::Facet* fpool,
                                      int* fids,
                                      int* fsizesSum,
                                      int* stOffsets,
                                      int* offsets,
                                      char* buffer,
                                      int num) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < num) {
        MCGAL::Facet* facet = &fpool[fids[tid]];
        // 需要知道自己在自己这个mesh中是第几个
        int offset = stOffsets[facet->meshId] + tid - fsizesSum[facet->meshId];
        char symbol = readCharOnCuda(buffer, offset);
        if (symbol) {
            facet->setSplittableOnCuda();
            offsets[tid] = 1;
        } else {
            facet->setUnsplittableOnCuda();
            offsets[tid] = 0;
        }
    }
}

__global__ void
readPointOnCuda(MCGAL::Facet* fpool, int* fids, int* fsizesSum, int* stOffsets, int* offsets, char* buffer, int num) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < num) {
        MCGAL::Facet* facet = &fpool[fids[tid]];
        if (facet->isSplittableOnCuda()) {
            int st = facet->meshId == 0 ? 0 : offsets[fsizesSum[facet->meshId] - 1];
            int offset = offsets[tid] - 1 - st;
            float* p = readPointOnCuda(buffer, stOffsets[facet->meshId] + offset * sizeof(float) * 3);
            facet->setRemovedVertexPosOnCuda(p);
        }
    }
}

__global__ void
readHalfedgeSymbolOnCuda(MCGAL::Halfedge* hpool, int* hids, int* hsizesSum, int* stOffsets, char* buffer, int num) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < num) {
        MCGAL::Halfedge* halfedge = &hpool[hids[tid]];
        // 需要知道自己在自己这个mesh中是第几个
        int offset = stOffsets[halfedge->meshId] + tid - hsizesSum[halfedge->meshId];
        char symbol = readCharOnCuda(buffer, offset);
        if (symbol) {
            halfedge->setAddedOnCuda();
        }
    }
}

__global__ void arrayAdd(int* arr1, int* arr2, int num) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < num) {
        arr1[tid] = arr1[tid] + arr2[tid];
    }
}

__global__ void calSplitableCounts(int* stOffsets, int* splitableCounts, int* offsets, int* fsizesSum, int num) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < num) {
        int ed = offsets[fsizesSum[tid + 1] - 1];
        int st = fsizesSum[tid] == 0 ? 0 : offsets[fsizesSum[tid] - 1];
        splitableCounts[tid] = ed - st;
        stOffsets[tid] += (splitableCounts[tid]) * sizeof(float) * 3;
    }
}

__global__ void initFsizesSum(int* fsizesSum, int* fsizes, int index, int batch_size) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid == 0) {
        for (int i = 1; i < batch_size; i++) {
            fsizesSum[i] = fsizesSum[i - 1] + fsizes[i];
        }
        fsizesSum[batch_size] = index;
    }
}

__global__ void initHsizesSum(int* hsizesSum, int* hsizes, int index, int batch_size) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid == 0) {
        for (int i = 1; i < batch_size; i++) {
            hsizesSum[i] = hsizesSum[i - 1] + hsizes[i];
        }
        hsizesSum[batch_size] = index;
    }
}

__global__ void checkOffset(int* stOffsets, int batch_size) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < batch_size) {
        if (stOffsets[tid] % 4 != 0) {
            stOffsets[tid] = (stOffsets[tid] / 4 + 1) * 4;
        }
    }
}

void DeCompressTool::BatchRemovedVerticesDecodingStep() {
    struct timeval start = get_cur_time();
    int size = *MCGAL::contextPool.findex;
    thrust::device_vector<int> origin_fids(size);
    thrust::device_vector<int> fids(size);
    // 使用 thrust::transform 提取facet中的 poolId
    thrust::transform(MCGAL::contextPool.dfpool, MCGAL::contextPool.dfpool + size, origin_fids.begin(),
                      ExtractFacetPoolId());
    // 仅拷贝meshId不为1的部分
    thrust::copy_if(origin_fids.begin(), origin_fids.end(), fids.begin(),
                    FilterFacetByMeshId(MCGAL::contextPool.dfpool));
    // 获取紧凑后的数组大小
    int index = thrust::count_if(thrust::device, origin_fids.begin(), origin_fids.end(),
                                 FilterFacetByMeshId(MCGAL::contextPool.dfpool));
    logt("%d thrust init in remove vertex", start, i_curDecimationId);

    // 初始化每个面的数量以及前缀和
    int* fsizes;
    int* hfsizes = new int[batch_size];
    memset(hfsizes, 0, batch_size * sizeof(int));
    int* fsizesSum;
    int* hfsizesSum = new int[batch_size + 1];
    memset(hfsizesSum, 0, (batch_size + 1) * sizeof(int));
    CHECK(hipMalloc(&fsizes, batch_size * sizeof(int)));
    CHECK(hipMalloc(&fsizesSum, (batch_size + 1) * sizeof(int)));
    CHECK(hipMemcpy(fsizes, hfsizes, batch_size * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(fsizesSum, hfsizesSum, (batch_size + 1) * sizeof(int), hipMemcpyHostToDevice));
    dim3 block(256, 1, 1);
    dim3 grid((index + block.x - 1) / block.x, 1, 1);
    // 统计每个mesh中面的数量，方便之后计算offset
    countFacetOccurrences<<<grid, block>>>(MCGAL::contextPool.dfpool, thrust::raw_pointer_cast(fids.data()), fsizes,
                                           index);
    hipDeviceSynchronize();
    hipMemcpy(hfsizes, fsizes, batch_size * sizeof(int), hipMemcpyDeviceToHost);

    // 为fsizes计算前缀和
    initFsizesSum<<<1, 1>>>(fsizesSum, fsizes, index, batch_size);
    hipDeviceSynchronize();
    // 检查offset是否为4的倍数
    checkOffset<<<1, batch_size>>>(dstOffsets, batch_size);
    hipDeviceSynchronize();
    int* d_firstQueue;
    int* d_secondQueue;
    int* d_nextQueueSize;
    int nextQueueSize = 0;
    CHECK(hipMalloc((void**)&d_firstQueue, size));
    CHECK(hipMalloc((void**)&d_secondQueue, size));
    CHECK(hipMalloc((void**)&d_nextQueueSize, sizeof(int)));
    CHECK(hipMemcpy(d_nextQueueSize, &nextQueueSize, sizeof(int), hipMemcpyHostToDevice));
    int* h_firstQueue = new int[batch_size];
    int* stIds = new int[batch_size];
    int* dstIds;
    CHECK(hipMalloc((void**)&dstIds, batch_size * sizeof(int)));
    int currentQueueSize = batch_size;

    int level = 0;
    for (int i = 0; i < batch_size; i++) {
        MCGAL::Halfedge* hit = pushHehInit(i);
        // hit->facet()->forder = 0;
        stIds[i] = hit->facet_;
        h_firstQueue[i] = hit->poolId;
    }

    CHECK(hipMemcpy(dstIds, stIds, batch_size * sizeof(int), hipMemcpyHostToDevice));
    // set forder by cuda
    initForder<<<1, batch_size>>>(MCGAL::contextPool.dfpool, dstIds, batch_size);
    hipDeviceSynchronize();
    // copy first to queue
    CHECK(hipMemcpy(d_firstQueue, h_firstQueue, batch_size * sizeof(int), hipMemcpyHostToDevice));
    logt("%d bfs init in remove vertex", start, i_curDecimationId);

    int threshold = 64 / 4 - 1;
    int firstCount = 0;
    int secondCount = 0;
    while (currentQueueSize > 0) {
        int* d_currentQueue;
        int* d_nextQueue;
        if (level % 2 == 0) {
            d_currentQueue = d_firstQueue;
            d_nextQueue = d_secondQueue;
        } else {
            d_currentQueue = d_secondQueue;
            d_nextQueue = d_firstQueue;
        }
        dim3 block(256, 1, 1);
        dim3 grid((currentQueueSize + block.x - 1) / block.x, 1, 1);
        computeFacetNextQueue<<<grid, block>>>(MCGAL::contextPool.dvpool, MCGAL::contextPool.dhpool,
                                               MCGAL::contextPool.dfpool, d_currentQueue, d_nextQueue, d_nextQueueSize,
                                               currentQueueSize);
        hipDeviceSynchronize();
        ++level;
        // 到达阈值后开始compact
        if (level == threshold) {
            struct timeval compact = get_cur_time();
            // 需要一个新的临时的array，以order进行排序
            thrust::sort(fids.begin() + firstCount, fids.begin() + index, SortFacetByForder(MCGAL::contextPool.dfpool));

            secondCount += thrust::count_if(thrust::device, fids.begin() + firstCount, fids.begin() + index,
                                            FilterFacetByForder(MCGAL::contextPool.dfpool));
            thrust::device_vector<int> incId(secondCount - firstCount);
            thrust::sequence(incId.begin(), incId.end());
            thrust::for_each(thrust::device, incId.begin(), incId.end(),
                             UpdateFacetOrderFunctor(thrust::raw_pointer_cast(fids.data()) + firstCount,
                                                     MCGAL::contextPool.dfpool, firstCount));
            firstCount = secondCount;
            int power = 1;
            int x = secondCount + 1;
            while (x > 1) {
                x /= 2;
                power++;
            }
            threshold += (64 - power) / 4 - 1;
            logt("%d %d compact", compact, i_curDecimationId, level);
        }

        setProcessedProcessedFlagOnCuda<<<grid, block>>>(MCGAL::contextPool.dhpool, MCGAL::contextPool.dfpool,
                                                         d_currentQueue, currentQueueSize);
        hipDeviceSynchronize();
        CHECK(hipMemcpy(&currentQueueSize, d_nextQueueSize, sizeof(int), hipMemcpyDeviceToHost));
        // currentQueueSize = nextQueueSize;
        CHECK(hipMemcpy(d_nextQueueSize, &nextQueueSize, sizeof(int), hipMemcpyHostToDevice));
    }
    logt("%d bfs", start, i_curDecimationId);
    // sort
    // sort(fids, fids + index, cmpForder);
    thrust::sort(fids.begin(), fids.begin() + index, SortFacetByMeshId(MCGAL::contextPool.dfpool));
    logt("%d sort", start, i_curDecimationId);
    int* hoffset = new int[index];
    memset(hoffset, 0, sizeof(int) * index);
    int* d_offset;

    // 需要计算前缀和
    CHECK(hipMalloc(&d_offset, index * sizeof(int)));
    readFacetSymbolOnCuda<<<grid, block>>>(MCGAL::contextPool.dfpool, thrust::raw_pointer_cast(fids.data()), fsizesSum,
                                           dstOffsets, d_offset, dbuffer, index);
    hipDeviceSynchronize();
    CHECK(hipMemcpy(hfsizesSum, fsizesSum, (batch_size + 1) * sizeof(int), hipMemcpyDeviceToHost));

    // 求前缀和，用于计算offset
    thrust::inclusive_scan(thrust::device, d_offset, d_offset + index, d_offset);
    arrayAdd<<<1, batch_size>>>(dstOffsets, fsizes, batch_size);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("ERROR: %s:%d,", __FILE__, __LINE__);
        printf("code:%d,reason:%s\n", error, hipGetErrorString(error));
        exit(1);
    }
    // 检查offset是否为4的倍数
    checkOffset<<<1, batch_size>>>(dstOffsets, batch_size);
    hipDeviceSynchronize();
    // 根据offset的值读取point
    readPointOnCuda<<<grid, block>>>(MCGAL::contextPool.dfpool, thrust::raw_pointer_cast(fids.data()), fsizesSum,
                                     dstOffsets, d_offset, dbuffer, index);
    hipDeviceSynchronize();
    // 检查offset是否为4的倍数
    checkOffset<<<1, batch_size>>>(dstOffsets, batch_size);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("ERROR: %s:%d,", __FILE__, __LINE__);
        printf("code:%d,reason:%s\n", error, hipGetErrorString(error));
        exit(1);
    }
    // 计算splitableCount
    calSplitableCounts<<<1, batch_size>>>(dstOffsets, dSplittabelCount, d_offset, fsizesSum, batch_size);
    hipDeviceSynchronize();

    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("ERROR: %s:%d,", __FILE__, __LINE__);
        printf("code:%d,reason:%s\n", error, hipGetErrorString(error));
        exit(1);
    }

    // int vsize = *MCGAL::contextPool.vindex;
    // int hsize = *MCGAL::contextPool.hindex;
    // int fsize = *MCGAL::contextPool.findex;
    // CHECK(hipMemcpy(MCGAL::contextPool.vpool, MCGAL::contextPool.dvpool, vsize * sizeof(MCGAL::Vertex),
    //                  hipMemcpyDeviceToHost));
    // CHECK(hipMemcpy(MCGAL::contextPool.hpool, MCGAL::contextPool.dhpool, hsize * sizeof(MCGAL::Halfedge),
    //                  hipMemcpyDeviceToHost));
    // CHECK(hipMemcpy(MCGAL::contextPool.fpool, MCGAL::contextPool.dfpool, fsize * sizeof(MCGAL::Facet),
    //                  hipMemcpyDeviceToHost));
    // CHECK(hipMemcpy(stOffsets.data(), dstOffsets, sizeof(int) * batch_size, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(splitableCounts.data(), dSplittabelCount, sizeof(int) * batch_size, hipMemcpyDeviceToHost));
}

void DeCompressTool::BatchInsertedEdgeDecodingStepOnCuda() {
    struct timeval start = get_cur_time();

    int size = *MCGAL::contextPool.hindex;
    thrust::device_vector<int> origin_hids(size);
    thrust::device_vector<int> hids(size);
    // 使用 thrust::transform 提取facet中的 poolId
    thrust::transform(MCGAL::contextPool.dhpool, MCGAL::contextPool.dhpool + size, origin_hids.begin(),
                      ExtractHalfedgePoolId());
    // 仅拷贝meshId不为1的部分
    thrust::copy_if(origin_hids.begin(), origin_hids.end(), hids.begin(),
                    FilterHalfedgeByMeshId(MCGAL::contextPool.dhpool));
    // 获取紧凑后的数组大小
    int index = thrust::count_if(thrust::device, origin_hids.begin(), origin_hids.end(),
                                 FilterHalfedgeByMeshId(MCGAL::contextPool.dhpool));
    // 初始化每个面的数量以及前缀和
    int* hsizes;
    int* hhsizes = new int[batch_size];
    memset(hhsizes, 0, batch_size * sizeof(int));
    int* hsizesSum;
    int* hhsizesSum = new int[batch_size + 1];
    memset(hhsizesSum, 0, (batch_size + 1) * sizeof(int));
    CHECK(hipMalloc(&hsizes, batch_size * sizeof(int)));
    CHECK(hipMalloc(&hsizesSum, (batch_size + 1) * sizeof(int)));
    CHECK(hipMemcpy(hsizes, hhsizes, batch_size * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(hsizesSum, hhsizesSum, (batch_size + 1) * sizeof(int), hipMemcpyHostToDevice));
    dim3 block(256, 1, 1);
    dim3 grid((index + block.x - 1) / block.x, 1, 1);
    // 统计每个mesh中面的数量，方便之后计算offset
    countHalfedgeOccurrences<<<grid, block>>>(MCGAL::contextPool.dhpool, thrust::raw_pointer_cast(hids.data()), hsizes,
                                              index);
    hipDeviceSynchronize();
    hipMemcpy(hhsizes, hsizes, batch_size * sizeof(int), hipMemcpyDeviceToHost);

    // 为hsizes计算前缀和
    initHsizesSum<<<1, 1>>>(hsizesSum, hsizes, index, batch_size);
    hipDeviceSynchronize();
    logt("%d thrust init in remove vertex", start, i_curDecimationId);

    int* d_firstQueue;
    int* d_secondQueue;
    int* d_nextQueueSize;
    int nextQueueSize = 0;
    CHECK(hipMalloc((void**)&d_firstQueue, size));
    CHECK(hipMalloc((void**)&d_secondQueue, size));
    CHECK(hipMalloc((void**)&d_nextQueueSize, sizeof(int)));
    CHECK(hipMemcpy(d_nextQueueSize, &nextQueueSize, sizeof(int), hipMemcpyHostToDevice));
    int* h_firstQueue = new int[batch_size];
    int currentQueueSize = batch_size;
    int level = 0;
    for (int i = 0; i < batch_size; i++) {
        MCGAL::Halfedge* hit = pushHehInit(i);
        h_firstQueue[i] = hit->poolId;
    }

    // copy first to queue
    CHECK(hipMemcpy(d_firstQueue, h_firstQueue, batch_size * sizeof(int), hipMemcpyHostToDevice));
    // set forder by cuda
    initHorder<<<1, batch_size>>>(MCGAL::contextPool.dhpool, d_firstQueue, batch_size);
    hipDeviceSynchronize();
    int threshold = 64 / 4 - 1;
    int firstCount = 0;
    int secondCount = 0;
    while (currentQueueSize > 0) {
        int* d_currentQueue;
        int* d_nextQueue;
        if (level % 2 == 0) {
            d_currentQueue = d_firstQueue;
            d_nextQueue = d_secondQueue;
        } else {
            d_currentQueue = d_secondQueue;
            d_nextQueue = d_firstQueue;
        }
        dim3 block(256, 1, 1);
        dim3 grid((currentQueueSize + block.x - 1) / block.x, 1, 1);
        computeHalfedgeNextQueue<<<grid, block>>>(MCGAL::contextPool.dvpool, MCGAL::contextPool.dhpool,
                                                  MCGAL::contextPool.dfpool, d_currentQueue, d_nextQueue,
                                                  d_nextQueueSize, currentQueueSize);
        hipDeviceSynchronize();
        ++level;
        // 到达阈值后开始compact
        if (level == threshold) {
            struct timeval compact = get_cur_time();
            thrust::sort(hids.begin() + firstCount, hids.begin() + index,
                         SortHalfedgeByHorder(MCGAL::contextPool.dhpool));

            secondCount += thrust::count_if(thrust::device, hids.begin() + firstCount, hids.begin() + index,
                                            FilterHalfedgeByHorder(MCGAL::contextPool.dhpool));
            thrust::device_vector<int> incId(secondCount - firstCount);
            thrust::sequence(incId.begin(), incId.end());
            thrust::for_each(thrust::device, incId.begin(), incId.end(),
                             UpdateHalfedgeOrderFunctor(thrust::raw_pointer_cast(hids.data()) + firstCount,
                                                        MCGAL::contextPool.dhpool, firstCount));

            firstCount = secondCount;
            int power = 1;
            int x = secondCount + 1;
            while (x > 1) {
                x /= 2;
                power++;
            }
            threshold += (64 - power) / 4 - 1;
            logt("%d level %d compact", compact, i_curDecimationId, level);
        }

        CHECK(hipMemcpy(&currentQueueSize, d_nextQueueSize, sizeof(int), hipMemcpyDeviceToHost));
        // currentQueueSize = nextQueueSize;
        CHECK(hipMemcpy(d_nextQueueSize, &nextQueueSize, sizeof(int), hipMemcpyHostToDevice));
    }
    logt("%d bfs", start, i_curDecimationId);
    thrust::sort(hids.begin(), hids.begin() + index, SortHalfedgeByMeshId(MCGAL::contextPool.dhpool));

    readHalfedgeSymbolOnCuda<<<grid, block>>>(MCGAL::contextPool.dhpool, thrust::raw_pointer_cast(hids.data()),
                                              hsizesSum, dstOffsets, dbuffer, index);
    hipDeviceSynchronize();
    arrayAdd<<<1, batch_size>>>(dstOffsets, hsizes, batch_size);
    hipDeviceSynchronize();
    checkOffset<<<1, batch_size>>>(dstOffsets, batch_size);
    hipDeviceSynchronize();

    // int vsize = *MCGAL::contextPool.vindex;
    // int hsize = *MCGAL::contextPool.hindex;
    // int fsize = *MCGAL::contextPool.findex;
    // CHECK(hipMemcpy(MCGAL::contextPool.vpool, MCGAL::contextPool.dvpool, vsize * sizeof(MCGAL::Vertex),
    //                  hipMemcpyDeviceToHost));
    // CHECK(hipMemcpy(MCGAL::contextPool.hpool, MCGAL::contextPool.dhpool, hsize * sizeof(MCGAL::Halfedge),
    //                  hipMemcpyDeviceToHost));
    // CHECK(hipMemcpy(MCGAL::contextPool.fpool, MCGAL::contextPool.dfpool, fsize * sizeof(MCGAL::Facet),
    //                  hipMemcpyDeviceToHost));
}

void DeCompressTool::BatchInsertedEdgeDecodingStep() {
    int size = *MCGAL::contextPool.hindex;
    int* firstQueue = new int[size];
    int* secondQueue = new int[size];
    int currentQueueSize = batch_size;
    int nextQueueSize = 0;
    int level = 0;
    int threshold = 64 / 4 - 1;
    int firstCount = 0;
    int secondCount = 0;
    int* hids = new int[size];
    // 将信息拷贝过来
    int index = 0;
    int* hsizes = new int[batch_size];
    int* hsizesSum = new int[batch_size + 1];
    memset(hsizes, 0, batch_size * sizeof(int));
    memset(hsizesSum, 0, (batch_size + 1) * sizeof(int));
    for (int i = 0; i < size; i++) {
        MCGAL::Halfedge* halfedge = &MCGAL::contextPool.hpool[i];
        if (halfedge->meshId != -1) {
            hids[index++] = i;
            hsizes[halfedge->facet()->meshId]++;
        }
    }
    for (int i = 1; i < batch_size; i++) {
        hsizesSum[i] = hsizesSum[i - 1] + hsizes[i];
    }
    hsizesSum[batch_size] = index;

    for (int i = 0; i < batch_size; i++) {
        MCGAL::Halfedge* hit = pushHehInit(i);
        hit->horder = 0;
        firstQueue[i] = hit->poolId;
    }

    while (currentQueueSize > 0) {
        int* currentQueue;
        int* nextQueue;
        if (level % 2 == 0) {
            currentQueue = firstQueue;
            nextQueue = secondQueue;
        } else {
            currentQueue = secondQueue;
            nextQueue = firstQueue;
        }
#pragma omp parallel for num_threads(128)
        for (int i = 0; i < currentQueueSize; i++) {
            int current = currentQueue[i];
            MCGAL::Halfedge* h = MCGAL::contextPool.getHalfedgeByIndex(current);
            if (h->isProcessed()) {
                continue;
            }
            MCGAL::Halfedge* hIt = h->next();
            unsigned long long idx = 1;
            while (hIt->opposite() != h) {
                unsigned long long order = h->horder << 4 | idx;

#pragma omp atomic compare
                hIt->horder = order < hIt->horder ? order : hIt->horder;

                if (hIt->horder == order) {
                    idx++;
                    int position;
#pragma omp critical
                    { position = nextQueueSize++; }
                    nextQueue[position] = hIt->poolId;
                }
                hIt = hIt->opposite()->next();
            };
        }
        ++level;
        // 到达阈值后开始compact
        if (level == threshold) {
            // sort(halfedges.begin() + firstCount, halfedges.begin() + secondCount, cmpHorder);
            sort(hids, hids + index, cmpHorder);

            for (int i = firstCount; i < index; i++) {
                if (MCGAL::contextPool.hpool[hids[i]].horder != (~(unsigned long long)0)) {
                    MCGAL::contextPool.hpool[hids[i]].horder = i;
                    secondCount = i;
                }
            }
            firstCount = secondCount;

            int power = 1;
            int x = secondCount + 1;
            while (x > 1) {
                x /= 2;
                power++;
            }
            threshold += (64 - power) / 4 - 1;
        }
        // offFile << "\n";
        currentQueueSize = nextQueueSize;
        nextQueueSize = 0;
    }
    sort(hids, hids + index, cmpHorder);
    // 并行读取
#pragma omp parallel for num_threads(128)
    for (int i = 0; i < index; i++) {
        MCGAL::Halfedge* halfedge = MCGAL::contextPool.getHalfedgeByIndex(hids[i]);
        int offset = stOffsets[halfedge->meshId] + i - hsizesSum[halfedge->meshId];
        char symbol = readCharByOffset(offset);
        if (symbol) {
            halfedge->setAdded();
        }
    }
    for (int i = 0; i < batch_size; i++) {
        stOffsets[i] += hsizes[i];
        if (stOffsets[i] % 4 != 0) {
            stOffsets[i] = (stOffsets[i] / 4 + 1) * 4;
        }
    }
    hipMemcpy(dstOffsets, stOffsets.data(), batch_size * sizeof(int), hipMemcpyHostToDevice);
    delete firstQueue;
    delete secondQueue;
}

void DeCompressTool::RemovedVerticesDecodingStep(int meshId) {
    std::queue<MCGAL::Halfedge*> gateQueue;
    int splitable_count = 0;
    gateQueue.push(pushHehInit(meshId));
    while (!gateQueue.empty()) {
        MCGAL::Halfedge* h = gateQueue.front();
        gateQueue.pop();

        MCGAL::Facet* f = h->facet();

        // If the face is already processed, pick the next halfedge:
        if (f->isConquered())
            continue;

        // Add the other halfedges to the queue
        MCGAL::Halfedge* hIt = h;
        do {
            MCGAL::Halfedge* hOpp = hIt->opposite();
            // TODO: wait
            // assert(!hOpp->is_border());
            if (!hOpp->facet()->isConquered())
                gateQueue.push(hOpp);
            hIt = hIt->next();
        } while (hIt != h);

        // Decode the face symbol.
        unsigned sym = readChar(&stOffsets[meshId]);
        if (sym == 1) {
            MCGAL::Point rmved = readPoint(&stOffsets[meshId]);
            f->setSplittable();
            splitable_count++;
            f->setRemovedVertexPos(rmved);
        } else {
            f->setUnsplittable();
        }
    }
    splitableCounts[meshId] = splitable_count;
}

void DeCompressTool::InsertedEdgeDecodingStep(int meshId) {
    std::queue<MCGAL::Halfedge*> gateQueue;
    int inserted_edgecount = 0;
    gateQueue.push(pushHehInit(meshId));
    while (!gateQueue.empty()) {
        MCGAL::Halfedge* h = gateQueue.front();
        gateQueue.pop();

        // Test if the edge has already been conquered.
        if (h->isProcessed())
            continue;

        // Mark the halfedge as processed.
        h->setProcessed();
        // h->opposite()->setProcessed();

        unsigned sym = readChar(&stOffsets[meshId]);
        // Determine if the edge is original or not.
        // Mark the edge to be removed.
        if (sym != 0) {
            h->setAdded();
            inserted_edgecount++;
        }

        // Add the other halfedges to the queue
        MCGAL::Halfedge* hIt = h->next();
        while (hIt->opposite() != h) {
            if (!hIt->isProcessed() && !hIt->isNew())
                gateQueue.push(hIt);
            hIt = hIt->opposite()->next();
        }
        assert(!hIt->isNew());
    }
}

inline __device__ void insert_tip_cuda(MCGAL::Halfedge* hs, MCGAL::Halfedge* h, MCGAL::Halfedge* v) {
    h->setNextOnCuda(v->dnext(hs));
    v->setNextOnCuda(h->dopposite(hs));
}

// kernel function
__global__ void createCenterVertexOnCuda(MCGAL::Vertex* vpool,
                                         MCGAL::Halfedge* hpool,
                                         MCGAL::Facet* fpool,
                                         int* vertexIndexes,
                                         int* faceIndexes,
                                         int* stHalfedgeIndexes,
                                         int* stFacetIndexes,
                                         int num,
                                         double clockRate,
                                         int id) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < num) {
        unsigned int startTime = clock64();

        int faceId = faceIndexes[tid];
        MCGAL::Facet* facet = &fpool[faceId];
        int vertexId = vertexIndexes[tid];
        MCGAL::Vertex* vnew = &vpool[vertexId];
        int stHalfedgeIndex = stHalfedgeIndexes[tid];
        int stFacetIndex = stFacetIndexes[tid];

        MCGAL::Halfedge* h = facet->getHalfedgeByIndexOnCuda(hpool, 0);
        MCGAL::Halfedge* hnew = &hpool[stHalfedgeIndex++];
        hnew->resetOnCuda(h->dend_vertex(vpool), vnew);

        MCGAL::Halfedge* oppo_new = &hpool[stHalfedgeIndex++];
        oppo_new->resetOnCuda(vnew, h->dend_vertex(vpool));
        hnew->setOppositeOnCuda(oppo_new);
        oppo_new->setOppositeOnCuda(hnew);
        insert_tip_cuda(hpool, hnew->dopposite(hpool), h);
        MCGAL::Halfedge* g = hnew->dopposite(hpool)->dnext(hpool);
        MCGAL::Halfedge* hed = hnew;
        while (g->dnext(hpool)->poolId != hed->poolId) {
            MCGAL::Halfedge* gnew = &hpool[stHalfedgeIndex++];
            gnew->resetOnCuda(g->dend_vertex(vpool), vnew);

            MCGAL::Halfedge* oppo_gnew = &hpool[stHalfedgeIndex++];
            oppo_gnew->resetOnCuda(vnew, g->dend_vertex(vpool));

            gnew->setOppositeOnCuda(oppo_gnew);
            oppo_gnew->setOppositeOnCuda(gnew);
            gnew->setNextOnCuda(hnew->dopposite(hpool));
            insert_tip_cuda(hpool, gnew->dopposite(hpool), g);
            g = gnew->dopposite(hpool)->dnext(hpool);
            hnew = gnew;
        }

        hed->setNextOnCuda(hnew->dopposite(hpool));
        for (int i = 1; i < h->dfacet(fpool)->halfedge_size; i += 1) {
            MCGAL::Halfedge* hit = &hpool[h->dfacet(fpool)->halfedges[i]];
            fpool[stFacetIndex++].resetOnCuda(vpool, hpool, hit);
        }
        h->dfacet(fpool)->resetOnCuda(vpool, hpool, h);
    }
}

__device__ int allocateVertexFromPool(MCGAL::Vertex* vpool, float* p, int* vindex) {
    int tpIndex = atomicAdd(vindex, 1);
    vpool[tpIndex].setPointOnCuda(p);
    return tpIndex;
}

/**
 * 先算stFaceIndex
 * 再算stHalfedgeIndex
 * 算vertexIndex，不需要求前缀和了，直接加上索引即可
 * 先设置那些数组的值，然后求前缀和，然后全部加一个值
 * 难点是在 vertex addHalfedge
 * 最好是以vertex为单位来进行，因为不会存在竞争
 * 让每个新加进来halfedge知道自己要进入到哪个vertex中
 */

__global__ void initStIndexes(MCGAL::Vertex* vpool,
                              MCGAL::Halfedge* hpool,
                              MCGAL::Facet* fpool,
                              int* vertexIndexes,
                              int* faceIndexes,
                              int* stFacetIndexes,
                              int* stHalfedgeIndexes,
                              int num) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < num) {
        MCGAL::Facet* fit = &fpool[faceIndexes[tid]];
        int hcount = fit->halfedge_size * 2;
        int fcount = fit->halfedge_size - 1;
        vertexIndexes[tid] = 1;
        stFacetIndexes[tid] = fcount;
        stHalfedgeIndexes[tid] = hcount;
    }
}

__global__ void arrayAddConstant(int* array, int constant, int num) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < num) {
        array[tid] = array[tid] + constant;
    }
}

__global__ void preAllocInit(MCGAL::Vertex* vpool,
                             MCGAL::Halfedge* hpool,
                             MCGAL::Facet* fpool,
                             int* vertexIndexes,
                             int* faceIndexes,
                             int* stFacetIndexes,
                             int* stHalfedgeIndexes,
                             int num) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < num) {
        MCGAL::Facet* fit = &fpool[faceIndexes[tid]];

        int hcount = fit->halfedge_size * 2;
        int fcount = fit->halfedge_size - 1;
        int stfindex = stFacetIndexes[tid];
        for (int i = 0; i < fcount; i++) {
            fpool[stfindex + i].setMeshIdOnCuda(fit->meshId);
        }
        int stHindex = stHalfedgeIndexes[tid];
        for (int j = 0; j < hcount; j++) {
            hpool[stHindex + j].setMeshIdOnCuda(fit->meshId);
        }
        MCGAL::Vertex* vnew = &vpool[vertexIndexes[tid]];
        vnew->setMeshIdOnCuda(fit->meshId);
        vnew->setPointOnCuda(fit->getRemovedVertexPosOnCuda());
    }
}

// __global__ void preAllocPostProcessor(MCGAL::Vertex* vpool,
//                                       MCGAL::Halfedge* hpool,
//                                       MCGAL::Facet* fpool,
//                                       int* vertexIndexes,
//                                       int* faceIndexes,
//                                       int* stFacetIndexes,
//                                       int* stHalfedgeIndexes,
//                                       int num) {
//     int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
//     if (tid < num) {
//         MCGAL::Facet* fit = &fpool[faceIndexes[tid]];

//         int hcount = fit->halfedge_size * 2;
//         int fcount = fit->halfedge_size - 1;
//         int stfindex = stFacetIndexes[tid];
//         for (int i = 0; i < fcount; i++) {
//             fpool[stfindex + i].setMeshIdOnCuda(fit->meshId);
//         }
//         int stHindex = stHalfedgeIndexes[tid];
//         for (int j = 0; j < hcount; j++) {
//             hpool[stHindex + j].setMeshIdOnCuda(fit->meshId);
//         }
//         MCGAL::Vertex* vnew = &vpool[vertexIndexes[tid]];
//         vnew->setMeshIdOnCuda(fit->meshId);
//     }
// }

__global__ void preAllocOnCuda(MCGAL::Vertex* vpool,
                               MCGAL::Halfedge* hpool,
                               MCGAL::Facet* fpool,
                               int* findex,
                               int* hindex,
                               int* vindex,
                               int* vertexIndexes,
                               int* faceIndexes,
                               int* stFacetIndexes,
                               int* stHalfedgeIndexes,
                               int num) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < num) {
        MCGAL::Facet* fit = &fpool[faceIndexes[tid]];
        int hcount = fit->halfedge_size * 2;
        int fcount = fit->halfedge_size - 1;
        int stfindex = atomicAdd(findex, fcount);
        for (int i = 0; i < fcount; i++) {
            fpool[stfindex + i].setMeshIdOnCuda(fit->meshId);
        }
        stFacetIndexes[tid] = stfindex;
        int stHindex = atomicAdd(hindex, hcount);
        stHalfedgeIndexes[tid] = stHindex;
        for (int j = 0; j < hcount; j++) {
            hpool[stHindex + j].setMeshIdOnCuda(fit->meshId);
        }
        int ret = allocateVertexFromPool(vpool, fit->getRemovedVertexPosOnCuda(), vindex);
        // printf("%d ", ret);
        MCGAL::Vertex* vnew = &vpool[ret];
        vertexIndexes[tid] = vnew->poolId;
        vnew->setMeshIdOnCuda(fit->meshId);
        vnew->setPointOnCuda(fit->getRemovedVertexPosOnCuda());
        // for (int k = 0; k < fit->halfedge_size; k++) {
        //     MCGAL::Halfedge* h = &hpool[fit->halfedges[k]];
        //     h->dend_vertex(vpool)->addHalfedgeOnCuda(stHindex + k * 2);
        //     vnew->addHalfedgeOnCuda(stHindex + k * 2 + 1);
        // }
    }
}

// void DeCompressTool::insertRemovedVerticesOnCuda() {
//     struct timeval start = get_cur_time();
//     int size = *MCGAL::contextPool.findex;
//     thrust::device_vector<int> origin_fids(size);
//     int splitable_count = 0;
//     for (int i = 0; i < splitableCounts.size(); i++) {
//         splitable_count += splitableCounts[i];
//     }
//     thrust::device_vector<int> faceIndexes(size);
//     thrust::device_vector<int> vertexIndexes(size);
//     thrust::device_vector<int> stHalfedgeIndexes(size);
//     thrust::device_vector<int> stFacetIndexes(size);
//     hipError_t error = hipGetLastError();
//     if (error != hipSuccess) {
//         printf("ERROR: %s:%d,", __FILE__, __LINE__);
//         printf("code:%d,reason:%s\n", error, hipGetErrorString(error));
//         exit(1);
//     }
//     // 使用 thrust::transform 提取facet中的 poolId
//     thrust::transform(MCGAL::contextPool.dfpool, MCGAL::contextPool.dfpool + size, origin_fids.begin(),
//                       ExtractFacetPoolId());
//     // 拷贝所有splittable的facet
//     thrust::copy_if(origin_fids.begin(), origin_fids.end(), faceIndexes.begin(),
//                     FilterFacetBySplitable(MCGAL::contextPool.dfpool));
//     // 获取紧凑后的数组大小
//     int index = thrust::count_if(thrust::device, origin_fids.begin(), origin_fids.end(),
//                                  FilterFacetBySplitable(MCGAL::contextPool.dfpool));
//     error = hipGetLastError();
//     if (error != hipSuccess) {
//         printf("ERROR: %s:%d,", __FILE__, __LINE__);
//         printf("code:%d,reason:%s\n", error, hipGetErrorString(error));
//         exit(1);
//     }
//     logt("%d thrust init in real remove vertex", start, i_curDecimationId);
//     log("index:%d,splittable:%d", index, splitable_count);
//     dim3 block(128, 1, 1);
//     dim3 grid((splitable_count + block.x - 1) / block.x, 1, 1);
//     preAllocOnCuda<<<grid, block>>>(
//         MCGAL::contextPool.dvpool, MCGAL::contextPool.dhpool, MCGAL::contextPool.dfpool, MCGAL::contextPool.findex,
//         MCGAL::contextPool.hindex, MCGAL::contextPool.vindex, thrust::raw_pointer_cast(vertexIndexes.data()),
//         thrust::raw_pointer_cast(faceIndexes.data()), thrust::raw_pointer_cast(stFacetIndexes.data()),
//         thrust::raw_pointer_cast(stHalfedgeIndexes.data()), splitable_count);
//     hipDeviceSynchronize();
//     logt("%d prealloc on cuda", start, i_curDecimationId);
//     error = hipGetLastError();
//     if (error != hipSuccess) {
//         printf("ERROR: %s:%d,", __FILE__, __LINE__);
//         printf("code:%d,reason:%s\n", error, hipGetErrorString(error));
//         exit(1);
//     }
//     hipDeviceProp_t prop;
//     hipGetDeviceProperties(&prop, 0);
//     double clockRate = prop.clockRate;
//     createCenterVertexOnCuda<<<grid, block>>>(
//         MCGAL::contextPool.dvpool, MCGAL::contextPool.dhpool, MCGAL::contextPool.dfpool,
//         thrust::raw_pointer_cast(vertexIndexes.data()), thrust::raw_pointer_cast(faceIndexes.data()),
//         thrust::raw_pointer_cast(stHalfedgeIndexes.data()), thrust::raw_pointer_cast(stFacetIndexes.data()),
//         splitable_count, clockRate, i_curDecimationId);
//     hipDeviceSynchronize();
//     logt("%d core kernel", start, i_curDecimationId);
//     error = hipGetLastError();
//     if (error != hipSuccess) {
//         printf("ERROR: %s:%d,", __FILE__, __LINE__);
//         printf("code:%d,reason:%s\n", error, hipGetErrorString(error));
//         exit(1);
//     }
//     int vsize = *MCGAL::contextPool.vindex;
//     int hsize = *MCGAL::contextPool.hindex;
//     int fsize = *MCGAL::contextPool.findex;
//     CHECK(hipMemcpy(MCGAL::contextPool.vpool, MCGAL::contextPool.dvpool, vsize * sizeof(MCGAL::Vertex),
//                      hipMemcpyDeviceToHost));
//     CHECK(hipMemcpy(MCGAL::contextPool.hpool, MCGAL::contextPool.dhpool, hsize * sizeof(MCGAL::Halfedge),
//                      hipMemcpyDeviceToHost));
//     CHECK(hipMemcpy(MCGAL::contextPool.fpool, MCGAL::contextPool.dfpool, fsize * sizeof(MCGAL::Facet),
//                      hipMemcpyDeviceToHost));
// }

void DeCompressTool::insertRemovedVerticesOnCuda() {
    struct timeval start = get_cur_time();
    int size = *MCGAL::contextPool.findex;
    thrust::device_vector<int> origin_fids(size);

    int splitable_count = 0;
    for (int i = 0; i < splitableCounts.size(); i++) {
        splitable_count += splitableCounts[i];
    }
    thrust::device_vector<int> faceIndexes(splitable_count);
    thrust::device_vector<int> vertexIndexes(splitable_count + 1);
    vertexIndexes[0] = 0;
    thrust::device_vector<int> stHalfedgeIndexes(splitable_count + 1);
    stHalfedgeIndexes[0] = 0;
    thrust::device_vector<int> stFacetIndexes(splitable_count + 1);
    stFacetIndexes[0] = 0;
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("ERROR: %s:%d,", __FILE__, __LINE__);
        printf("code:%d,reason:%s\n", error, hipGetErrorString(error));
        exit(1);
    }
    // 使用 thrust::transform 提取facet中的 poolId
    thrust::transform(MCGAL::contextPool.dfpool, MCGAL::contextPool.dfpool + size, origin_fids.begin(),
                      ExtractFacetPoolId());
    // 拷贝所有splittable的facet
    thrust::copy_if(origin_fids.begin(), origin_fids.end(), faceIndexes.begin(),
                    FilterFacetBySplitable(MCGAL::contextPool.dfpool));
    // 获取紧凑后的数组大小
    int index = thrust::count_if(thrust::device, origin_fids.begin(), origin_fids.end(),
                                 FilterFacetBySplitable(MCGAL::contextPool.dfpool));
    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("ERROR: %s:%d,", __FILE__, __LINE__);
        printf("code:%d,reason:%s\n", error, hipGetErrorString(error));
        exit(1);
    }
    logt("%d thrust init in real remove vertex", start, i_curDecimationId);
    dim3 block(128, 1, 1);
    dim3 grid((splitable_count + block.x - 1) / block.x, 1, 1);
    initStIndexes<<<grid, block>>>(MCGAL::contextPool.dvpool, MCGAL::contextPool.dhpool, MCGAL::contextPool.dfpool,
                                   thrust::raw_pointer_cast(vertexIndexes.data()) + 1,
                                   thrust::raw_pointer_cast(faceIndexes.data()),
                                   thrust::raw_pointer_cast(stFacetIndexes.data()) + 1,
                                   thrust::raw_pointer_cast(stHalfedgeIndexes.data()) + 1, splitable_count);
    hipDeviceSynchronize();
    thrust::inclusive_scan(thrust::device, vertexIndexes.begin(), vertexIndexes.end(), vertexIndexes.begin());
    thrust::inclusive_scan(thrust::device, stFacetIndexes.begin(), stFacetIndexes.end(), stFacetIndexes.begin());
    thrust::inclusive_scan(thrust::device, stHalfedgeIndexes.begin(), stHalfedgeIndexes.end(),
                           stHalfedgeIndexes.begin());
    arrayAddConstant<<<grid, block>>>(thrust::raw_pointer_cast(vertexIndexes.data()), *MCGAL::contextPool.vindex,
                                      splitable_count);
    hipDeviceSynchronize();
    arrayAddConstant<<<grid, block>>>(thrust::raw_pointer_cast(stHalfedgeIndexes.data()), *MCGAL::contextPool.hindex,
                                      splitable_count);
    hipDeviceSynchronize();
    arrayAddConstant<<<grid, block>>>(thrust::raw_pointer_cast(stFacetIndexes.data()), *MCGAL::contextPool.findex,
                                      splitable_count);
    hipDeviceSynchronize();
    preAllocInit<<<grid, block>>>(MCGAL::contextPool.dvpool, MCGAL::contextPool.dhpool, MCGAL::contextPool.dfpool,
                                  thrust::raw_pointer_cast(vertexIndexes.data()),
                                  thrust::raw_pointer_cast(faceIndexes.data()),
                                  thrust::raw_pointer_cast(stFacetIndexes.data()),
                                  thrust::raw_pointer_cast(stHalfedgeIndexes.data()), splitable_count);
    hipDeviceSynchronize();
    int vindex;
    int hindex;
    int findex;
    hipMemcpy(&vindex, thrust::raw_pointer_cast(vertexIndexes.data()) + splitable_count, sizeof(int),
               hipMemcpyDeviceToHost);
    hipMemcpy(&hindex, thrust::raw_pointer_cast(stHalfedgeIndexes.data()) + splitable_count, sizeof(int),
               hipMemcpyDeviceToHost);
    hipMemcpy(&findex, thrust::raw_pointer_cast(stFacetIndexes.data()) + splitable_count, sizeof(int),
               hipMemcpyDeviceToHost);
    *MCGAL::contextPool.vindex += vindex;
    *MCGAL::contextPool.hindex += hindex;
    *MCGAL::contextPool.findex += findex;
    // preAllocOnCuda<<<grid, block>>>(
    //     MCGAL::contextPool.dvpool, MCGAL::contextPool.dhpool, MCGAL::contextPool.dfpool, MCGAL::contextPool.findex,
    //     MCGAL::contextPool.hindex, MCGAL::contextPool.vindex, thrust::raw_pointer_cast(vertexIndexes.data()),
    //     thrust::raw_pointer_cast(faceIndexes.data()), thrust::raw_pointer_cast(stFacetIndexes.data()),
    //     thrust::raw_pointer_cast(stHalfedgeIndexes.data()), splitable_count);
    logt("%d prealloc on cuda", start, i_curDecimationId);
    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("ERROR: %s:%d,", __FILE__, __LINE__);
        printf("code:%d,reason:%s\n", error, hipGetErrorString(error));
        exit(1);
    }
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    double clockRate = prop.clockRate;
    createCenterVertexOnCuda<<<grid, block>>>(
        MCGAL::contextPool.dvpool, MCGAL::contextPool.dhpool, MCGAL::contextPool.dfpool,
        thrust::raw_pointer_cast(vertexIndexes.data()), thrust::raw_pointer_cast(faceIndexes.data()),
        thrust::raw_pointer_cast(stHalfedgeIndexes.data()), thrust::raw_pointer_cast(stFacetIndexes.data()),
        splitable_count, clockRate, i_curDecimationId);
    hipDeviceSynchronize();
    logt("%d core kernel", start, i_curDecimationId);
    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("ERROR: %s:%d,", __FILE__, __LINE__);
        printf("code:%d,reason:%s\n", error, hipGetErrorString(error));
        exit(1);
    }
    int vsize = *MCGAL::contextPool.vindex;
    int hsize = *MCGAL::contextPool.hindex;
    int fsize = *MCGAL::contextPool.findex;
    CHECK(hipMemcpy(MCGAL::contextPool.vpool, MCGAL::contextPool.dvpool, vsize * sizeof(MCGAL::Vertex),
                     hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(MCGAL::contextPool.hpool, MCGAL::contextPool.dhpool, hsize * sizeof(MCGAL::Halfedge),
                     hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(MCGAL::contextPool.fpool, MCGAL::contextPool.dfpool, fsize * sizeof(MCGAL::Facet),
                     hipMemcpyDeviceToHost));
}

// 多线程预处理
// 将最后一步放到cuda上
void DeCompressTool::insertRemovedVertices() {
    struct timeval start = get_cur_time();
    int splitable_count = 0;
    for (int i = 0; i < splitableCounts.size(); i++) {
        splitable_count += splitableCounts[i];
    }

    std::vector<int> faceIndexes(splitable_count);
    std::vector<int> vertexIndexes(splitable_count);
    std::vector<int> stHalfedgeIndexes(splitable_count);
    std::vector<int> stFacetIndexes(splitable_count);
    int index = 0;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    double clockRate = prop.clockRate;
    int findex = *MCGAL::contextPool.findex;
//
#pragma omp parallel for num_threads(50) schedule(dynamic)
    for (int i = 0; i < findex; i++) {
        MCGAL::Facet* fit = MCGAL::contextPool.getFacetByIndex(i);
        if (fit->meshId != -1 && fit->isSplittable()) {
            faceIndexes[index] = fit->poolId;
            int hcount = fit->halfedge_size * 2;
            int fcount = fit->halfedge_size - 1;
            // atomic
            int findex;
#pragma omp critical
            { findex = MCGAL::contextPool.preAllocFace(fcount); }
            for (int i = 0; i < fcount; i++) {
                MCGAL::contextPool.getFacetByIndex(findex + i)->setMeshId(fit->meshId);
            }
            stFacetIndexes[index] = findex;
            int hindex;
#pragma omp critical
            {
                hindex = MCGAL::contextPool.preAllocHalfedge(hcount);
                stHalfedgeIndexes[index] = hindex;
            }
            for (int j = 0; j < hcount; j++) {
                MCGAL::Halfedge* h = MCGAL::contextPool.getHalfedgeByIndex(hindex + j);
                h->setMeshId(fit->meshId);
            }
            MCGAL::Vertex* vnew;
#pragma omp critical
            {
                vertexIndexes[index] = MCGAL::contextPool.getVindex();
                // atomic
                vnew = MCGAL::contextPool.allocateVertexFromPool(fit->getRemovedVertexPos());
            }
            vnew->setMeshId(fit->meshId);
#pragma omp atomic
            index++;
            for (int k = 0; k < fit->halfedge_size; k++) {
                MCGAL::Halfedge* h = fit->getHalfedgeByIndex(k);
                h->end_vertex()->addHalfedge(hindex + k * 2);
                vnew->addHalfedge(hindex + k * 2 + 1);
            }
        }
    }
    // add it to mesh
    int num = splitable_count;
    dim3 block(256, 1, 1);
    dim3 grid((num + block.x - 1) / block.x, 1, 1);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("ERROR: %s:%d,", __FILE__, __LINE__);
        printf("code:%d,reason:%s\n", error, hipGetErrorString(error));
        exit(1);
    }
    CHECK(hipMemcpy(dfaceIndexes, faceIndexes.data(), faceIndexes.size() * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dvertexIndexes, vertexIndexes.data(), vertexIndexes.size() * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dstHalfedgeIndexes, stHalfedgeIndexes.data(), stHalfedgeIndexes.size() * sizeof(int),
                     hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dstFacetIndexes, stFacetIndexes.data(), stFacetIndexes.size() * sizeof(int),
                     hipMemcpyHostToDevice));
    int vsize = *MCGAL::contextPool.vindex;
    int hsize = *MCGAL::contextPool.hindex;
    int fsize = *MCGAL::contextPool.findex;
    // log("size is %d %d %d", vsize, hsize, fsize);
    CHECK(hipMemcpy(MCGAL::contextPool.dvpool, MCGAL::contextPool.vpool, vsize * sizeof(MCGAL::Vertex),
                     hipMemcpyHostToDevice));
    CHECK(hipMemcpy(MCGAL::contextPool.dhpool, MCGAL::contextPool.hpool, hsize * sizeof(MCGAL::Halfedge),
                     hipMemcpyHostToDevice));
    CHECK(hipMemcpy(MCGAL::contextPool.dfpool, MCGAL::contextPool.fpool, fsize * sizeof(MCGAL::Facet),
                     hipMemcpyHostToDevice));
    createCenterVertexOnCuda<<<grid, block>>>(MCGAL::contextPool.dvpool, MCGAL::contextPool.dhpool,
                                              MCGAL::contextPool.dfpool, dvertexIndexes, dfaceIndexes,
                                              dstHalfedgeIndexes, dstFacetIndexes, num, clockRate, i_curDecimationId);
    hipDeviceSynchronize();
    double t = logt("%d kernel function", start, i_curDecimationId);
    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("ERROR: %s:%d,", __FILE__, __LINE__);
        printf("code:%d,reason:%s\n", error, hipGetErrorString(error));
        exit(1);
    }
    CHECK(hipMemcpy(MCGAL::contextPool.vpool, MCGAL::contextPool.dvpool, vsize * sizeof(MCGAL::Vertex),
                     hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(MCGAL::contextPool.hpool, MCGAL::contextPool.dhpool, hsize * sizeof(MCGAL::Halfedge),
                     hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(MCGAL::contextPool.fpool, MCGAL::contextPool.dfpool, fsize * sizeof(MCGAL::Facet),
                     hipMemcpyDeviceToHost));
    logt("%d cuda memory copy back", start, i_curDecimationId);
    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("ERROR: %s:%d,", __FILE__, __LINE__);
        printf("code:%d,reason:%s\n", error, hipGetErrorString(error));
        exit(1);
    }
}

__device__ MCGAL::Halfedge* find_prevOncuda(MCGAL::Halfedge* hpool, MCGAL::Halfedge* h) {
    MCGAL::Halfedge* g = h;
    int idx = 0;
    while (g->dnext(hpool) != h) {
        if (idx >= 120) {
            printf("error\n");
            break;
        }
        idx++;
        g = g->dnext(hpool);
    }

    return g;
}

inline __device__ void remove_tipOnCuda(MCGAL::Halfedge* hpool, MCGAL::Halfedge* h) {
    // h->next = h->next->opposite->next;
    h->setNextOnCuda(h->dnext(hpool)->dopposite(hpool)->dnext(hpool));
}

__device__ void joinFacetDevice(MCGAL::Vertex* vpool, MCGAL::Halfedge* hpool, MCGAL::Facet* fpool, MCGAL::Halfedge* h) {
    MCGAL::Halfedge* hprev = find_prevOncuda(hpool, h);
    MCGAL::Halfedge* gprev = find_prevOncuda(hpool, h->dopposite(hpool));
    atomicAdd(&h->count, 1);
    // atomicAdd(&hprev->count, 1);
    remove_tipOnCuda(hpool, hprev);
    remove_tipOnCuda(hpool, gprev);
    gprev->dfacet(fpool)->setRemovedOnCuda();
    hprev->dfacet(fpool)->resetOnCuda(vpool, hpool, hprev);
}

__global__ void joinFacetOnCuda(MCGAL::Vertex* vpool,
                                MCGAL::Halfedge* hpool,
                                MCGAL::Facet* fpool,
                                int* edgeIndexes,
                                int* stIndexes,
                                int* thNumberes,
                                int num,
                                double clockRate) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < num) {
        int stIndex = stIndexes[tid];
        int thNumber = thNumberes[tid];
        for (int i = 0; i < thNumber; i++) {
            MCGAL::Halfedge* h = &hpool[edgeIndexes[stIndex + i]];
            joinFacetDevice(vpool, hpool, fpool, h);
        }
    }
}

MCGAL::Halfedge* DeCompressTool::find_prev(MCGAL::Halfedge* h) const {
    MCGAL::Halfedge* g = h;
    while (g->next() != h)
        g = g->next();
    return g;
}

inline void DeCompressTool::remove_tip(MCGAL::Halfedge* h) const {
    h->next_ = h->next()->opposite()->next_;
}

__global__ void initParent(MCGAL::Halfedge* hpool, MCGAL::Facet* fpool, int* hids, int num) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < num) {
        MCGAL::Halfedge* halfedge = &hpool[tid];
        halfedge->parent = min(halfedge->poolId, halfedge->opposite_);
    }
}
// 尝试两边同时find

__global__ void mergeParent(MCGAL::Halfedge* hpool, MCGAL::Facet* fpool, int* hids, int num) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < num) {
        MCGAL::Halfedge* halfedge = &hpool[tid];
        if (halfedge->parent != halfedge->poolId) {
            MCGAL::Halfedge* tp = halfedge;
            while (true) {
                MCGAL::Halfedge* parent = &hpool[tp->parent];
                tp->parent = parent->parent;
                if (parent->poolId == parent->parent) {
                    break;
                }
            }
        }
    }
}

// 每个人都先找一下自己的parent

/**
 * Remove all the marked edges on cuda
 */
/**
 * 分为两步进行，第一步先标记，第二步合并
 */
// void DeCompressTool::removeInsertedEdgesOnCuda() {
//     int size = *MCGAL::contextPool.hindex;
//     thrust::device_vector<int> origin_hids(size);
//     thrust::device_vector<int> hids(size);
//     // 使用 thrust::transform 提取facet中的 poolId
//     thrust::transform(MCGAL::contextPool.dhpool, MCGAL::contextPool.dhpool + size, origin_hids.begin(),
//                       ExtractHalfedgePoolId());
//     // 仅拷贝meshId不为1的部分
//     thrust::copy_if(origin_hids.begin(), origin_hids.end(), hids.begin(),
//                     FilterHalfedgeByAdded(MCGAL::contextPool.dhpool));
//     // 获取紧凑后的数组大小
//     int index = thrust::count_if(thrust::device, origin_hids.begin(), origin_hids.end(),
//                                  FilterHalfedgeByAdded(MCGAL::contextPool.dhpool));
//     dim3 block(256, 1, 1);
//     dim3 grid((index + block.x - 1) / block.x, 1, 1);
//     // 初始化自己的parent
//     initParent<<<grid, block>>>(MCGAL::contextPool.dhpool, MCGAL::contextPool.dfpool,
//                                 thrust::raw_pointer_cast(hids.data()), index);
//     // 合并parent
//     mergeParent<<<grid, block>>>(MCGAL::contextPool.dhpool, MCGAL::contextPool.dfpool,
//                                  thrust::raw_pointer_cast(hids.data()), index);
//     // 得到最终的可用序列，使用thrust::unique
//     thrust::sort(hids.begin(), hids.end());
//     // 在排序后的数组上应用unique操作，得到不重复的元素
//     thrust::device_vector<int> stHalfedge = hids;
//     auto new_end = thrust::unique(stHalfedge.begin(), stHalfedge.end());

//     // 调整新数组的大小，以便只包含不重复的元素
//     stHalfedge.resize(thrust::distance(stHalfedge.begin(), new_end));
//     int num = stHalfedge.size();

//     hipDeviceProp_t prop;
//     hipGetDeviceProperties(&prop, 0);
//     double clockRate = prop.clockRate;
//     struct timeval start = get_cur_time();
//     // std::vector<int> edgeIndex(inserted_edgecount);
//     std::vector<int> edgeIndexes;
//     std::vector<int> stIndex;
//     std::vector<int> thNumber;
//     for (int i = 0; i < faces.size(); i++) {
//         MCGAL::Facet* node = faces[i];
//         if (node->isVisited()) {
//             continue;
//         }
//         // 记录这一轮bfs所有可用的面
//         std::vector<int> ids;
//         std::queue<MCGAL::Facet*> fqueue;
//         fqueue.push(node);
//         while (!fqueue.empty()) {
//             MCGAL::Facet* fit = fqueue.front();
//             fqueue.pop();
//             if (fit->isVisited()) {
//                 continue;
//             }
//             fit->setVisitedFlag();
//             int flag = 0;
//             for (int j = 0; j < fit->halfedge_size; j++) {
//                 MCGAL::Halfedge* hit = fit->getHalfedgeByIndex(j);
//                 MCGAL::Facet* fit2 = hit->opposite()->facet();
//                 if (hit->isAdded() && !hit->isVisited()) {
//                     ids.push_back(hit->poolId);
//                     hit->setVisited();
//                     hit->opposite()->setRemoved();
//                     // fit2->setRemoved();
//                     hit->vertex()->eraseHalfedgeByPointer(hit);
//                     hit->opposite()->vertex()->eraseHalfedgeByPointer(hit->opposite());
//                     fqueue.push(fit2);
//                 } else if (hit->opposite()->isAdded() && !hit->opposite()->isVisited()) {
//                     ids.push_back(hit->poolId);
//                     hit->opposite()->setVisited();
//                     hit->setRemoved();
//                     // fit2->setRemoved();
//                     hit->vertex()->eraseHalfedgeByPointer(hit);
//                     hit->opposite()->vertex()->eraseHalfedgeByPointer(hit->opposite());
//                     fqueue.push(fit2);
//                 }
//             }
//         }
//         if (!ids.empty()) {
//             stIndex.push_back(edgeIndexes.size());
//             for (int j = 0; j < ids.size(); j++) {
//                 edgeIndexes.push_back(ids[j]);
//             }
//             thNumber.push_back(ids.size());
//         }
//     }
//     logt("%d collect halfedge information", start, i_curDecimationId);
//     int* dedgeIndexes;
//     int* dstIndex;
//     int* dthNumber;
//     std::vector<int> edgeIndexesCnt(inserted_edgecount, 0);
//     CHECK(hipMalloc(&dedgeIndexes, edgeIndexes.size() * sizeof(int)));
//     CHECK(hipMalloc(&dstIndex, stIndex.size() * sizeof(int)));
//     CHECK(hipMalloc(&dthNumber, thNumber.size() * sizeof(int)));
//     CHECK(hipMemcpy(dedgeIndexes, edgeIndexes.data(), edgeIndexes.size() * sizeof(int), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(dstIndex, stIndex.data(), stIndex.size() * sizeof(int), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(dthNumber, thNumber.data(), thNumber.size() * sizeof(int), hipMemcpyHostToDevice));
//     int vsize = MCGAL::contextPool.vindex;
//     int hsize = MCGAL::contextPool.hindex;
//     int fsize = MCGAL::contextPool.findex;
//     int num = stIndex.size();
//     CHECK(hipMemcpy(MCGAL::contextPool.dvpool, MCGAL::contextPool.vpool, vsize * sizeof(MCGAL::Vertex),
//                      hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(MCGAL::contextPool.dhpool, MCGAL::contextPool.hpool, hsize * sizeof(MCGAL::Halfedge),
//                      hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(MCGAL::contextPool.dfpool, MCGAL::contextPool.fpool, fsize * sizeof(MCGAL::Facet),
//                      hipMemcpyHostToDevice));
//     dim3 block(256, 1, 1);
//     dim3 grid((num + block.x - 1) / block.x, 1, 1);
//     logt("%d cuda memcpy copy", start, i_curDecimationId);
//     joinFacetOnCuda<<<grid, block>>>(MCGAL::contextPool.dvpool, MCGAL::contextPool.dhpool, MCGAL::contextPool.dfpool,
//                                      dedgeIndexes, dstIndex, dthNumber, num, clockRate);
//     hipDeviceSynchronize();
//     logt("%d join facet kernel", start, i_curDecimationId);
//     hipError_t error = hipGetLastError();
//     if (error != hipSuccess) {
//         printf("ERROR: %s:%d,", __FILE__, __LINE__);
//         printf("code:%d,reason:%s\n", error, hipGetErrorString(error));
//         exit(1);
//     }
//     CHECK(hipMemcpy(MCGAL::contextPool.vpool, MCGAL::contextPool.dvpool, vsize * sizeof(MCGAL::Vertex),
//                      hipMemcpyDeviceToHost));
//     CHECK(hipMemcpy(MCGAL::contextPool.hpool, MCGAL::contextPool.dhpool, hsize * sizeof(MCGAL::Halfedge),
//                      hipMemcpyDeviceToHost));
//     CHECK(hipMemcpy(MCGAL::contextPool.fpool, MCGAL::contextPool.dfpool, fsize * sizeof(MCGAL::Facet),
//                      hipMemcpyDeviceToHost));
//     hipFree(dedgeIndexes);
//     hipFree(dstIndex);
//     hipFree(dthNumber);
//     // exit(0);
//     return;
// }

void DeCompressTool::removeInsertedEdges(int meshId) {
    std::queue<MCGAL::Halfedge*> gateQueue;
    gateQueue.push(pushHehInit(meshId));
    while (!gateQueue.empty()) {
        MCGAL::Halfedge* h = gateQueue.front();
        gateQueue.pop();

        if (h->isVisited())
            continue;

        if (h->isRemoved()) {
            continue;
        }
        // Mark the face as processed.
        h->setVisited();

        // Add the other halfedges to the queue
        MCGAL::Halfedge* hIt = h;
        do {
            MCGAL::Halfedge* hOpp = hIt->opposite();
            // TODO: wait
            // assert(!hOpp->is_border());
            if (!hOpp->isVisited())
                gateQueue.push(hOpp);
            hIt = hIt->next();
        } while (hIt != h);

        if (hIt->isRemoved()) {
            hIt->setVisited();
            continue;
        }
        if (hIt->isAdded()) {
            join_facet(hIt);
            hIt->setVisited();
        }
    }
    return;
}

MCGAL::Halfedge* DeCompressTool::join_facet(MCGAL::Halfedge* h) {
    MCGAL::Halfedge* hprev = find_prev(h);
    MCGAL::Halfedge* gprev = find_prev(h->opposite());
    remove_tip(hprev);
    remove_tip(gprev);
    h->opposite()->setRemoved();
    h->setRemoved();
    h->setMeshId(-1);
    h->opposite()->setMeshId(-1);
    h->vertex()->eraseHalfedgeByPointer(h);
    h->opposite()->vertex()->eraseHalfedgeByPointer(h->opposite());
    gprev->facet()->setRemoved();
    hprev->facet()->reset(hprev);
    return hprev;
}

#include "hip/hip_runtime.h"
// #include "../MCGAL/Core_CUDA/global.cuh"
#include "himesh.cuh"
#include "util.h"
#include <map>
#include <nvToolsExt.h>
#include <omp.h>

void HiMesh::decode(int lod) {
    assert(lod >= 0 && lod <= 100);
    // assert(!this->is_compression_mode());
    if (lod < i_decompPercentage) {
        return;
    }
    i_decompPercentage = lod;
    b_jobCompleted = false;
    while (!b_jobCompleted) {
        startNextDecompresssionOp();
    }
}

void HiMesh::startNextDecompresssionOp() {
    // check if the target LOD is reached
    if (i_curDecimationId * 100.0 / i_nbDecimations >= i_decompPercentage) {
        if (i_curDecimationId == i_nbDecimations) {}
        b_jobCompleted = true;
        return;
    }
    std::vector<int> twos;
    // 1. reset the states. note that the states of the vertices need not to be reset
    for (auto fit = faces.begin(); fit != faces.end();) {
        if ((*fit)->isRemoved()) {
            fit = faces.erase(fit);
        } else {
            (*fit)->resetState();
            for (int i = 0; i < (*fit)->halfedge_size; i++) {
                (*fit)->getHalfedgeByIndex(i)->resetState();
            }
            fit++;
        }
    }
    cur_offset = 0;
    cur_total = 0;
    splitable_count = 0;
    inserted_edgecount = 0;
    i_curDecimationId++;  // increment the current decimation operation id.
    // 2. decoding the removed vertices and add to target facets
    struct timeval start = get_cur_time();
    RemovedVerticesDecodingStep();
    logt("%d RemovedVerticesDecodingStep", start, i_curDecimationId);
    // 3. decoding the inserted edge and marking the ones added
    InsertedEdgeDecodingStep();
    logt("%d InsertedEdgeDecodingStep", start, i_curDecimationId);
    // 4. truly insert the removed vertices
    // insertRemovedVertices();
    insertRemovedVerticesOnCuda();
    logt("%d insertRemovedVertices", start, i_curDecimationId);
    // 5. truly remove the added edges
    removeInsertedEdgesOnCuda();
    // removeInsertedEdges();
    logt("%d removeInsertedEdges", start, i_curDecimationId);
}

void HiMesh::readBaseMesh() {
    // read the number of level of detail
    i_nbDecimations = readuInt16();
    // set the mesh bounding box
    unsigned i_nbVerticesBaseMesh = readInt();
    unsigned i_nbFacesBaseMesh = readInt();

    std::deque<MCGAL::Point>* p_pointDeque = new std::deque<MCGAL::Point>();
    std::deque<uint32_t*>* p_faceDeque = new std::deque<uint32_t*>();
    // Read the vertex positions.
    for (unsigned i = 0; i < i_nbVerticesBaseMesh; ++i) {
        MCGAL::Point pos = readPoint();
        p_pointDeque->push_back(pos);
    }
    // read the face vertex indices
    for (unsigned i = 0; i < i_nbFacesBaseMesh; ++i) {
        int nv = readInt();
        uint32_t* f = new uint32_t[nv + 1];
        // Write in the first cell of the array the face degree.
        f[0] = nv;
        for (unsigned j = 1; j < nv + 1; ++j) {
            f[j] = readInt();
        }
        p_faceDeque->push_back(f);
    }
    // Let the builder do its job.
    buildFromBuffer(p_pointDeque, p_faceDeque);

    // Free the memory.
    for (unsigned i = 0; i < p_faceDeque->size(); ++i) {
        delete[] p_faceDeque->at(i);
    }
    delete p_faceDeque;
    delete p_pointDeque;
}

void HiMesh::buildFromBuffer(std::deque<MCGAL::Point>* p_pointDeque, std::deque<uint32_t*>* p_faceDeque) {
    this->vertices.clear();
    // this->halfedges.clear();
    // used to create faces
    std::vector<MCGAL::Vertex*> vertices;
    // add vertex to Mesh
    for (std::size_t i = 0; i < p_pointDeque->size(); ++i) {
        MCGAL::Point p = p_pointDeque->at(i);
        MCGAL::Vertex* vt = MCGAL::contextPool.allocateVertexFromPool(p);
        vt->setId(i);
        this->vertices.push_back(vt);
        vertices.push_back(vt);
    }
    this->vh_departureConquest[0] = vertices[0];
    this->vh_departureConquest[1] = vertices[1];
    // read face and add to Mesh
    for (int i = 0; i < p_faceDeque->size(); ++i) {
        uint32_t* ptr = p_faceDeque->at(i);
        int num_face_vertices = ptr[0];
        std::vector<MCGAL::Vertex*> vts;
        for (int j = 0; j < num_face_vertices; ++j) {
            int vertex_index = ptr[j + 1];
            vts.push_back(vertices[vertex_index]);
        }
        MCGAL::Facet* face = MCGAL::contextPool.allocateFaceFromPool(vts);
        this->add_face(face);
        // this->faces
    }
    // clear vector
    vertices.clear();
}

void HiMesh::RemovedVerticesDecodingStep() {
    // 首先读出totalOffset
    // 然后开始随机读取
    cur_total = readInt();
    pushHehInit();
    while (!gateQueue.empty()) {
        MCGAL::Halfedge* h = gateQueue.front();
        gateQueue.pop();

        MCGAL::Facet* f = h->facet();

        // If the face is already processed, pick the next halfedge:
        if (f->isConquered())
            continue;

        // Add the other halfedges to the queue
        MCGAL::Halfedge* hIt = h;
        do {
            MCGAL::Halfedge* hOpp = hIt->opposite();
            // TODO: wait
            // assert(!hOpp->is_border());
            if (!hOpp->facet()->isConquered())
                gateQueue.push(hOpp);
            hIt = hIt->next();
        } while (hIt != h);

        // Decode the face symbol.
        int offset = readInt() + cur_total;
        unsigned sym = readCharByOffset(offset);
        cur_offset += 1;
        if (sym == 1) {
            MCGAL::Point rmved = readPointByOffset(offset + 1);
            cur_offset += sizeof(float) * 3;
            f->setSplittable();
            splitable_count++;
            f->setRemovedVertexPos(rmved);
        } else {
            f->setUnsplittable();
        }
    }
    // dataOffset = before + cur_total + sizeof(int);
}

/**
 * One step of the inserted edge coding conquest.
 */
void HiMesh::InsertedEdgeDecodingStep() {
    // 首先读出totalOffset
    // 然后开始随机读取
    pushHehInit();
    while (!gateQueue.empty()) {
        MCGAL::Halfedge* h = gateQueue.front();
        gateQueue.pop();

        // Test if the edge has already been conquered.
        if (h->isProcessed())
            continue;

        // Mark the halfedge as processed.
        h->setProcessed();
        h->opposite()->setProcessed();

        int offset = readInt() + cur_total;
        unsigned sym = readCharByOffset(offset);
        cur_offset += 1;
        // Determine if the edge is original or not.
        // Mark the edge to be removed.
        if (sym != 0) {
            h->setAdded();
            inserted_edgecount++;
        }

        // Add the other halfedges to the queue
        MCGAL::Halfedge* hIt = h->next();
        while (hIt->opposite() != h) {
            if (!hIt->isProcessed() && !hIt->isNew())
                gateQueue.push(hIt);
            hIt = hIt->opposite()->next();
        }
        assert(!hIt->isNew());
    }
    dataOffset += cur_offset;
}

inline __device__ void insert_tip_cuda(MCGAL::Halfedge* hs, MCGAL::Halfedge* h, MCGAL::Halfedge* v) {
    h->setNextOnCuda(v->dnext(hs));
    v->setNextOnCuda(h->dopposite(hs));
}

// kernel function
__global__ void createCenterVertexOnCuda(MCGAL::Vertex* vpool,
                                         MCGAL::Halfedge* hpool,
                                         MCGAL::Facet* fpool,
                                         int* vertexIndexes,
                                         int* faceIndexes,
                                         int* stHalfedgeIndexes,
                                         int* stFacetIndexes,
                                         int num,
                                         double clockRate,
                                         int id) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < num) {
        unsigned int startTime = clock64();

        int faceId = faceIndexes[tid];
        MCGAL::Facet* facet = &fpool[faceId];
        int vertexId = vertexIndexes[tid];
        MCGAL::Vertex* vnew = &vpool[vertexId];
        int stHalfedgeIndex = stHalfedgeIndexes[tid];
        int stFacetIndex = stFacetIndexes[tid];

        MCGAL::Halfedge* h = facet->getHalfedgeByIndexOnCuda(hpool, 0);
        MCGAL::Halfedge* hnew = &hpool[stHalfedgeIndex++];
        hnew->resetOnCuda(h->dend_vertex(vpool), vnew);

        MCGAL::Halfedge* oppo_new = &hpool[stHalfedgeIndex++];
        oppo_new->resetOnCuda(vnew, h->dend_vertex(vpool));
        hnew->setOppositeOnCuda(oppo_new);
        oppo_new->setOppositeOnCuda(hnew);
        insert_tip_cuda(hpool, hnew->dopposite(hpool), h);
        MCGAL::Halfedge* g = hnew->dopposite(hpool)->dnext(hpool);
        MCGAL::Halfedge* hed = hnew;
        while (g->dnext(hpool)->poolId != hed->poolId) {
            MCGAL::Halfedge* gnew = &hpool[stHalfedgeIndex++];
            gnew->resetOnCuda(g->dend_vertex(vpool), vnew);

            MCGAL::Halfedge* oppo_gnew = &hpool[stHalfedgeIndex++];
            oppo_gnew->resetOnCuda(vnew, g->dend_vertex(vpool));

            gnew->setOppositeOnCuda(oppo_gnew);
            oppo_gnew->setOppositeOnCuda(gnew);
            gnew->setNextOnCuda(hnew->dopposite(hpool));
            insert_tip_cuda(hpool, gnew->dopposite(hpool), g);
            g = gnew->dopposite(hpool)->dnext(hpool);
            hnew = gnew;
        }

        hed->setNextOnCuda(hnew->dopposite(hpool));
        for (int i = 1; i < h->dfacet(fpool)->halfedge_size; i += 1) {
            MCGAL::Halfedge* hit = &hpool[h->dfacet(fpool)->halfedges[i]];
            fpool[stFacetIndex++].resetOnCuda(vpool, hpool, hit);
        }
        h->dfacet(fpool)->resetOnCuda(vpool, hpool, h);
    }
}

#ifndef UNIFIED
void HiMesh::insertRemovedVerticesOnCuda() {
    struct timeval start = get_cur_time();
    std::vector<int> faceIndexes(splitable_count);
    std::vector<int> vertexIndexes(splitable_count);
    std::vector<int> stHalfedgeIndexes(splitable_count);
    std::vector<int> stFacetIndexes(splitable_count);
    int index = 0;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    double clockRate = prop.clockRate;
    for (int i = 0; i < faces.size(); i++) {
        MCGAL::Facet* fit = faces[i];
        if (fit->isSplittable()) {
            faceIndexes[index] = fit->poolId;
            int hcount = fit->halfedge_size * 2;
            int fcount = fit->halfedge_size - 1;
            int findex = MCGAL::contextPool.preAllocFace(fcount);
            for (int i = 0; i < fcount; i++) {
                this->faces.push_back(MCGAL::contextPool.getFacetByIndex(findex + i));
            }
            stFacetIndexes[index] = findex;

            int hindex = MCGAL::contextPool.preAllocHalfedge(hcount);
            stHalfedgeIndexes[index] = hindex;
            vertexIndexes[index] = (MCGAL::contextPool.getVindex());
            MCGAL::Vertex* vnew = MCGAL::contextPool.allocateVertexFromPool(fit->getRemovedVertexPos());
            this->vertices.push_back(vnew);
            index++;
            for (int i = 0; i < fit->halfedge_size; i++) {
                MCGAL::Halfedge* h = fit->getHalfedgeByIndex(i);
                h->end_vertex()->addHalfedge(hindex + i * 2);
                vnew->addHalfedge(hindex + i * 2 + 1);
            }
        }
    }
    logt("%d collect face information", start, i_curDecimationId);
    // add it to mesh
    int num = faceIndexes.size();
    dim3 block(256, 1, 1);
    dim3 grid((num + block.x - 1) / block.x, 1, 1);

    CHECK(hipMemcpy(dfaceIndexes, faceIndexes.data(), num * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dvertexIndexes, vertexIndexes.data(), num * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dstHalfedgeIndexes, stHalfedgeIndexes.data(), num * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dstFacetIndexes, stFacetIndexes.data(), num * sizeof(int), hipMemcpyHostToDevice));
    int vsize = MCGAL::contextPool.vindex;
    int hsize = MCGAL::contextPool.hindex;
    int fsize = MCGAL::contextPool.findex;
    // log("size is %d %d %d", vsize, hsize, fsize);
    CHECK(hipMemcpy(MCGAL::contextPool.dvpool, MCGAL::contextPool.vpool, vsize * sizeof(MCGAL::Vertex),
                     hipMemcpyHostToDevice));
    CHECK(hipMemcpy(MCGAL::contextPool.dhpool, MCGAL::contextPool.hpool, hsize * sizeof(MCGAL::Halfedge),
                     hipMemcpyHostToDevice));
    CHECK(hipMemcpy(MCGAL::contextPool.dfpool, MCGAL::contextPool.fpool, fsize * sizeof(MCGAL::Facet),
                     hipMemcpyHostToDevice));

    logt("%d cuda memory copy", start, i_curDecimationId);

#    ifdef TEST
    if (i_curDecimationId == 2) {
        grid.x = 19;
#        ifdef GRID_SIZE
        grid.x = GRID_SIZE;
#        endif
        block.x = 512;
#        ifdef BLOCK_SIZE
        block.x = BLOCK_SIZE;
#        endif
        block.y = 1;
    }
#    endif
    createCenterVertexOnCuda<<<grid, block>>>(MCGAL::contextPool.dvpool, MCGAL::contextPool.dhpool,
                                              MCGAL::contextPool.dfpool, dvertexIndexes, dfaceIndexes,
                                              dstHalfedgeIndexes, dstFacetIndexes, num, clockRate, i_curDecimationId);
    hipDeviceSynchronize();
    double t = logt("%d kernel function", start, i_curDecimationId);
#    ifdef TEST
    if (i_curDecimationId == 2) {
#        if defined(GRID_SIZE) && defined(BLOCK_SIZE)
        printf("%d %d %lf \n", GRID_SIZE, BLOCK_SIZE, t);
#        endif
        exit(0);
    }
#    endif
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("ERROR: %s:%d,", __FILE__, __LINE__);
        printf("code:%d,reason:%s\n", error, hipGetErrorString(error));
        exit(1);
    }
    CHECK(hipMemcpy(MCGAL::contextPool.vpool, MCGAL::contextPool.dvpool, vsize * sizeof(MCGAL::Vertex),
                     hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(MCGAL::contextPool.hpool, MCGAL::contextPool.dhpool, hsize * sizeof(MCGAL::Halfedge),
                     hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(MCGAL::contextPool.fpool, MCGAL::contextPool.dfpool, fsize * sizeof(MCGAL::Facet),
                     hipMemcpyDeviceToHost));
    logt("%d cuda memory copy back", start, i_curDecimationId);
    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("ERROR: %s:%d,", __FILE__, __LINE__);
        printf("code:%d,reason:%s\n", error, hipGetErrorString(error));
        exit(1);
    }
}
#else

void HiMesh::insertRemovedVerticesOnCuda() {
    hipSetDevice(0);
    struct timeval start = get_cur_time();
    std::vector<int> faceIndexes(splitable_count);
    std::vector<int> vertexIndexes(splitable_count);
    std::vector<int> stHalfedgeIndexes(splitable_count);
    std::vector<int> stFacetIndexes(splitable_count);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    double clockRate = prop.clockRate;
    int index = 0;
    for (int i = 0; i < faces.size(); i++) {
        MCGAL::Facet* fit = faces[i];
        if (fit->isSplittable()) {
            faceIndexes[index] = fit->poolId;
            int hcount = fit->halfedge_size * 2;
            int fcount = fit->halfedge_size - 1;
            int findex = MCGAL::contextPool.preAllocFace(fcount);
            for (int i = 0; i < fcount; i++) {
                this->faces.push_back(MCGAL::contextPool.getFacetByIndex(findex + i));
            }
            stFacetIndexes[index] = findex;

            int hindex = MCGAL::contextPool.preAllocHalfedge(hcount);
            stHalfedgeIndexes[index] = hindex;
            vertexIndexes[index] = (MCGAL::contextPool.getVindex());
            MCGAL::Vertex* vnew = MCGAL::contextPool.allocateVertexFromPool(fit->getRemovedVertexPos());
            this->vertices.push_back(vnew);
            index++;
            for (int i = 0; i < fit->halfedge_size; i++) {
                MCGAL::Halfedge* h = fit->getHalfedgeByIndex(i);
                h->end_vertex()->addHalfedge(hindex + i * 2);
                vnew->addHalfedge(hindex + i * 2 + 1);
            }
        }
    }
    logt("%d collect face information", start, i_curDecimationId);
    // add it to mesh
    int num = faceIndexes.size();
    dim3 block(512, 1, 1);
    dim3 grid((num + block.x - 1) / block.x, 1, 1);

    CHECK(hipMemcpy(dfaceIndexes, faceIndexes.data(), num * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dvertexIndexes, vertexIndexes.data(), num * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dstHalfedgeIndexes, stHalfedgeIndexes.data(), num * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dstFacetIndexes, stFacetIndexes.data(), num * sizeof(int), hipMemcpyHostToDevice));

    logt("%d cuda memory copy", start, i_curDecimationId);
    if (i_curDecimationId == 2) {
        grid.x = 19;
#    ifdef GRID_SIZE
        grid.x = GRID_SIZE;
#    endif
        block.x = 512;
#    ifdef BLOCK_SIZE
        block.x = BLOCK_SIZE;
#    endif
        block.y = 1;
    }
    createCenterVertexOnCuda<<<grid, block>>>(MCGAL::contextPool.vpool, MCGAL::contextPool.hpool,
                                              MCGAL::contextPool.fpool, dvertexIndexes, dfaceIndexes,
                                              dstHalfedgeIndexes, dstFacetIndexes, num, clockRate, i_curDecimationId);
    hipDeviceSynchronize();
    double t = logt("%d kernel function", start, i_curDecimationId);
    if (i_curDecimationId == 2) {
#    if defined(GRID_SIZE) && defined(BLOCK_SIZE)
        printf("%d %d %lf \n", GRID_SIZE, BLOCK_SIZE, t);
#    endif
        exit(0);
    }
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("ERROR: %s:%d,", __FILE__, __LINE__);
        printf("code:%d,reason:%s\n", error, hipGetErrorString(error));
        exit(1);
    }
}
#endif

__device__ MCGAL::Halfedge* find_prevOncuda(MCGAL::Halfedge* hpool, MCGAL::Halfedge* h) {
    MCGAL::Halfedge* g = h;
    int idx = 0;
    while (g->dnext(hpool) != h) {
        if (idx >= 120) {
            printf("error\n");
            break;
        }
        idx++;
        g = g->dnext(hpool);
    }

    return g;
}

inline __device__ void remove_tipOnCuda(MCGAL::Halfedge* hpool, MCGAL::Halfedge* h) {
    // h->next = h->next->opposite->next;
    h->setNextOnCuda(h->dnext(hpool)->dopposite(hpool)->dnext(hpool));
}

__global__ void resetHalfedgeOnCuda(MCGAL::Vertex* vpool,
                                    MCGAL::Halfedge* hpool,
                                    MCGAL::Facet* fpool,
                                    int* edgeIndexes,
                                    int* edgeIndexesCnt,
                                    int num,
                                    double clockRate) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < num) {
        MCGAL::Halfedge* hprev = &hpool[edgeIndexes[tid]];
        hprev->dfacet(fpool)->resetOnCuda(vpool, hpool, hprev);
    }
}

__device__ void acquireLock(int* lock) {
    while (atomicExch(lock, 1) != 0)
        ;
    __syncthreads();  // Wait for all threads to acquire the lock
}

__device__ void releaseLock(int* lock) {
    __syncthreads();  // Wait for all threads to reach this point
    atomicExch(lock, 0);
}

// __global__ void joinFacetOnCuda(MCGAL::Vertex* vpool,
//                                 MCGAL::Halfedge* hpool,
//                                 MCGAL::Facet* fpool,
//                                 int* facetIndexes,
//                                 int* stIndexes,
//                                 int* thNumberes,
//                                 int num,
//                                 double clockRate) {
//     int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
//     if (tid < num) {
//         int stIndex = stIndexes[tid];
//         int thNumber = thNumberes[tid];
//         int idx = 0;
//         int edge[100];
//         for (int i = 0; i < thNumber; i++) {
//             MCGAL::Facet* fit = &fpool[facetIndexes[stIndex + i]];
//             for (int j = 0; j < fit->halfedge_size; j++) {
//                 if (hpool[fit->halfedges[j]].isAddedOnCuda()) {
//                     edge[idx++] = fit->halfedges[j];
//                 }
//             }
//         }
//         // printf("%d\t", idx);
//         for (int i = 0; i < idx; i++) {
//             MCGAL::Halfedge* h = &hpool[edge[i]];
//             MCGAL::Facet* lockFacet = h->dfacet(fpool);
//             // MCGAL::Facet* lockOppoFacet = h->dopposite(hpool)->dfacet(fpool);
//             MCGAL::Halfedge* hprev = find_prevOncuda(hpool, h);
//             MCGAL::Halfedge* gprev = find_prevOncuda(hpool, h->dopposite(hpool));
//             remove_tipOnCuda(hpool, hprev);
//             remove_tipOnCuda(hpool, gprev);
//             // int hnext = h->dopposite(hpool)->next_;
//             // int gnext = h->next_;
//             // hprev->next_ = hnext;
//             // gprev->next_ = gnext;
//             // gprev->dfacet(fpool)->setRemovedOnCuda();
//             lockFacet->resetOnCuda(vpool, hpool, hprev);
//             if (lockFacet->halfedge_size >= 50) {
//                 printf("%d\t", thNumber);
//             }
//         }
//     }
// }

__device__ void joinFacetDevice(MCGAL::Vertex* vpool, MCGAL::Halfedge* hpool, MCGAL::Facet* fpool, MCGAL::Halfedge* h) {
    MCGAL::Halfedge* hprev = find_prevOncuda(hpool, h);
    MCGAL::Halfedge* gprev = find_prevOncuda(hpool, h->dopposite(hpool));
    atomicAdd(&h->count, 1);
    // atomicAdd(&hprev->count, 1);
    remove_tipOnCuda(hpool, hprev);
    remove_tipOnCuda(hpool, gprev);
    gprev->dfacet(fpool)->setRemovedOnCuda();
    hprev->dfacet(fpool)->resetOnCuda(vpool, hpool, hprev);
}

__global__ void joinFacetOnCuda(MCGAL::Vertex* vpool,
                                MCGAL::Halfedge* hpool,
                                MCGAL::Facet* fpool,
                                int* edgeIndexes,
                                int* stIndexes,
                                int* thNumberes,
                                int num,
                                double clockRate) {
    int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
    if (tid < num) {
        int stIndex = stIndexes[tid];
        int thNumber = thNumberes[tid];
        for (int i = 0; i < thNumber; i++) {
            MCGAL::Halfedge* h = &hpool[edgeIndexes[stIndex + i]];
            // acquireLock(&h->lock);
            joinFacetDevice(vpool, hpool, fpool, h);
            // releaseLock(&h->lock);
            // MCGAL::Halfedge* hprev = find_prevOncuda(hpool, h);
            // MCGAL::Halfedge* gprev = find_prevOncuda(hpool, h->dopposite(hpool));
            // atomicAdd(&hprev->count, 1);
            // atomicAdd(&hprev->count, 1);
            // remove_tipOnCuda(hpool, hprev);
            // remove_tipOnCuda(hpool, gprev);
            // gprev->dfacet(fpool)->setRemovedOnCuda();
            // hprev->dfacet(fpool)->resetOnCuda(vpool, hpool, hprev);
        }
    }
}

// __global__ void joinFacetOnCuda(MCGAL::Vertex* vpool,
//                                 MCGAL::Halfedge* hpool,
//                                 MCGAL::Facet* fpool,
//                                 int* edgeIndexes,
//                                 int* stIndexes,
//                                 int* thNumberes,
//                                 int num,
//                                 double clockRate) {
//     int tid = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;
//     if (tid < num) {
//         int stIndex = stIndexes[tid];
//         int thNumber = thNumberes[tid];
//         for (int i = 0; i < thNumber; i++) {
//             MCGAL::Halfedge* h = &hpool[edgeIndexes[stIndex + i]];
//             // join_face(h);
//             MCGAL::Halfedge* hprev = find_prevOncuda(hpool, h);
//             MCGAL::Halfedge* gprev = find_prevOncuda(hpool, h->dopposite(hpool));
//             atomicAdd(&hprev->count, 1);
//             atomicAdd(&hprev->count, 1);
//             remove_tipOnCuda(hpool, hprev);
//             remove_tipOnCuda(hpool, gprev);
//             // h->dopposite(hpool)->setRemoved();
//             // int hnext = h->dopposite(hpool)->next_;
//             // int gnext = h->next_;
//             // hprev->next_ = hnext;
//             // gprev->next_ = gnext;
//             gprev->dfacet(fpool)->setRemovedOnCuda();
//             hprev->dfacet(fpool)->resetOnCuda(vpool, hpool, hprev);
//         }
//     }
// }
/**
 * Remove all the marked edges on cuda
 */
/**
 * 以面为单位进行bfs，将所有联通的面作为一个block进行处理
 */
// void HiMesh::removeInsertedEdgesOnCuda() {
//     hipDeviceProp_t prop;
//     hipGetDeviceProperties(&prop, 0);
//     double clockRate = prop.clockRate;
//     struct timeval start = get_cur_time();
//     std::vector<int> facetIndexes;
//     std::vector<int> stIndexes;
//     std::vector<int> thNumbers;
//     for (int i = 0; i < faces.size(); i++) {
//         MCGAL::Facet* node = faces[i];
//         if (node->isVisited()) {
//             continue;
//         }
//         // 记录这一轮bfs所有可用的面
//         std::vector<int> ids;
//         std::queue<MCGAL::Facet*> fqueue;
//         fqueue.push(node);
//         while (!fqueue.empty()) {
//             MCGAL::Facet* fit = fqueue.front();
//             fqueue.pop();
//             if (fit->isVisited()) {
//                 continue;
//             }
//             fit->setVisitedFlag();
//             ids.push_back(fit->poolId);
//             for (int j = 0; j < fit->halfedge_size; j++) {
//                 MCGAL::Halfedge* hit = fit->getHalfedgeByIndex(j);
//                 if (hit->isAdded() && !hit->isVisited()) {
//                     // ids.push_back(hit->poolId);
//                     hit->setVisited();
//                     hit->opposite()->setRemoved();
//                     MCGAL::Facet* fit2 = hit->opposite()->facet();
//                     fit2->setRemoved();
//                     hit->vertex()->eraseHalfedgeByPointer(hit);
//                     hit->opposite()->vertex()->eraseHalfedgeByPointer(hit->opposite());
//                     // 入队
//                     fqueue.push(fit2);
//                 }
//             }
//         }
//         if (!ids.empty()) {
//             stIndexes.push_back(facetIndexes.size());
//             for (int j = 0; j < ids.size(); j++) {
//                 facetIndexes.push_back(ids[j]);
//             }
//             thNumbers.push_back(ids.size());
//         }
//     }
//     int* dfacetIndexes;
//     int* dstIndexes;
//     int* dthNumbers;
//     CHECK(hipMalloc(&dfacetIndexes, facetIndexes.size() * sizeof(int)));
//     CHECK(hipMalloc(&dstIndexes, stIndexes.size() * sizeof(int)));
//     CHECK(hipMalloc(&dthNumbers, thNumbers.size() * sizeof(int)));
//     CHECK(hipMemcpy(dfacetIndexes, facetIndexes.data(), facetIndexes.size() * sizeof(int), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(dstIndexes, stIndexes.data(), stIndexes.size() * sizeof(int), hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(dthNumbers, thNumbers.data(), thNumbers.size() * sizeof(int), hipMemcpyHostToDevice));
//     int vsize = MCGAL::contextPool.vindex;
//     int hsize = MCGAL::contextPool.hindex;
//     int fsize = MCGAL::contextPool.findex;
//     int num = stIndexes.size();
//     CHECK(hipMemcpy(MCGAL::contextPool.dvpool, MCGAL::contextPool.vpool, vsize * sizeof(MCGAL::Vertex),
//                      hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(MCGAL::contextPool.dhpool, MCGAL::contextPool.hpool, hsize * sizeof(MCGAL::Halfedge),
//                      hipMemcpyHostToDevice));
//     CHECK(hipMemcpy(MCGAL::contextPool.dfpool, MCGAL::contextPool.fpool, fsize * sizeof(MCGAL::Facet),
//                      hipMemcpyHostToDevice));
//     dim3 block(256, 1, 1);
//     dim3 grid((num + block.x - 1) / block.x, 1, 1);
//     logt("%d cuda memcpy copy", start, i_curDecimationId);
//     joinFacetOnCuda<<<grid, block>>>(MCGAL::contextPool.dvpool, MCGAL::contextPool.dhpool, MCGAL::contextPool.dfpool,
//                                      dfacetIndexes, dstIndexes, dthNumbers, num, clockRate);
//     hipDeviceSynchronize();
//     logt("%d join facet kernel", start, i_curDecimationId);
//     hipError_t error = hipGetLastError();
//     if (error != hipSuccess) {
//         printf("ERROR: %s:%d,", __FILE__, __LINE__);
//         printf("code:%d,reason:%s\n", error, hipGetErrorString(error));
//         exit(1);
//     }
//     CHECK(hipMemcpy(MCGAL::contextPool.vpool, MCGAL::contextPool.dvpool, vsize * sizeof(MCGAL::Vertex),
//                      hipMemcpyDeviceToHost));
//     CHECK(hipMemcpy(MCGAL::contextPool.hpool, MCGAL::contextPool.dhpool, hsize * sizeof(MCGAL::Halfedge),
//                      hipMemcpyDeviceToHost));
//     CHECK(hipMemcpy(MCGAL::contextPool.fpool, MCGAL::contextPool.dfpool, fsize * sizeof(MCGAL::Facet),
//                      hipMemcpyDeviceToHost));
//     hipFree(dfacetIndexes);
//     hipFree(dstIndexes);
//     hipFree(dthNumbers);
// }

// void HiMesh::removeInsertedEdgesOnCuda() {
//     hipDeviceProp_t prop;
//     hipGetDeviceProperties(&prop, 0);
//     double clockRate = prop.clockRate;
//     struct timeval start = get_cur_time();
//     // inserted_edgecount数量应该会比facet大一点
//     // 记录三个数组，第一个是所有待处理的边id，类似一个pool
//     // 第二个每个thread的起始index
//     // 第三个每个thread需要处理的数量
//     // std::vector<int> edgeIndex(inserted_edgecount);
//     std::vector<int> edgeIndexes;
//     std::vector<int> stIndexes;
//     std::vector<int> thNumberes;
//     for (int i = 0; i < faces.size(); i++) {
//         MCGAL::Facet* node = faces[i];
//         if (node->isVisited()) {
//             continue;
//         }
//         // 记录这一轮bfs所有可用的面
//         std::vector<int> ids;
//         std::queue<MCGAL::Facet*> fqueue;
//         fqueue.push(node);
//         while (!fqueue.empty()) {
//             MCGAL::Facet* fit = fqueue.front();
//             fqueue.pop();
//             if (fit->isVisited()) {
//                 continue;
//             }
//             fit->setVisitedFlag();
//             for (int j = 0; j < fit->halfedge_size; j++) {
//                 MCGAL::Halfedge* hit = fit->getHalfedgeByIndex(j);
//                 MCGAL::Facet* fit2 = hit->opposite()->facet();
//                 if (hit->isAdded() && !hit->isVisited()) {
//                     // MCGAL::Facet* fit2 = hit->opposite()->facet();
//                     // edgeIndex[idx++] = hit->poolId;
//                     ids.push_back(hit->poolId);
//                     hit->setVisited();
//                     hit->opposite()->setRemoved();
//                     // fit2->setRemoved();
//                     hit->vertex()->eraseHalfedgeByPointer(hit);
//                     hit->opposite()->vertex()->eraseHalfedgeByPointer(hit->opposite());
//                     // fqueue.push(fit2);
//                     // 入队
//                 }
//                 fqueue.push(fit2);
//             }
//         }
//         if (!ids.empty()) {
//             stIndexes.push_back(edgeIndexes.size());
//             for (int j = 0; j < ids.size(); j++) {
//                 edgeIndexes.push_back(ids[j]);
//             }
//             thNumberes.push_back(ids.size());
//         }
//     }
//     int stIndex = stIndexes[0];
//     int thNumber = thNumberes[0];
//     logt("%d collect information", start, i_curDecimationId);
// #pragma omp parallel for num_threads(60)
//     for (int i = 0; i < thNumber; i++) {
//         MCGAL::Halfedge* h = &MCGAL::contextPool.hpool[edgeIndexes[stIndex + i]];
//         // join_face(h);
//         MCGAL::Halfedge* hprev = find_prev(h);
//         MCGAL::Halfedge* gprev = find_prev(h->opposite());
//         int hnext = h->opposite()->next_;
//         int gnext = h->next_;
//         hprev->next_ = hnext;
//         gprev->next_ = gnext;
//         // remove_tip(hprev);
//         // remove_tip(gprev);
//         h->opposite()->setRemoved();
//         gprev->facet()->setRemoved();
//         hprev->facet()->reset(hprev);
//     }
//     logt("%d parallel", start, i_curDecimationId);
//     return;
// }

/**
 * Remove all the marked edges on cuda
 */
void HiMesh::removeInsertedEdgesOnCuda() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    double clockRate = prop.clockRate;
    struct timeval start = get_cur_time();
    // inserted_edgecount数量应该会比facet大一点
    // 记录三个数组，第一个是所有待处理的边id，类似一个pool
    // 第二个每个thread的起始index
    // 第三个每个thread需要处理的数量
    // std::vector<int> edgeIndex(inserted_edgecount);
    std::vector<int> edgeIndexes;
    std::vector<int> stIndex;
    std::vector<int> thNumber;
    for (int i = 0; i < faces.size(); i++) {
        MCGAL::Facet* node = faces[i];
        if (node->isVisited()) {
            continue;
        }
        // 记录这一轮bfs所有可用的面
        std::vector<int> ids;
        std::queue<MCGAL::Facet*> fqueue;
        fqueue.push(node);
        while (!fqueue.empty()) {
            MCGAL::Facet* fit = fqueue.front();
            fqueue.pop();
            if (fit->isVisited()) {
                continue;
            }
            fit->setVisitedFlag();
            int flag = 0;
            for (int j = 0; j < fit->halfedge_size; j++) {
                MCGAL::Halfedge* hit = fit->getHalfedgeByIndex(j);
                MCGAL::Facet* fit2 = hit->opposite()->facet();
                if (hit->isAdded() && !hit->isVisited()) {
                    // MCGAL::Facet* fit2 = hit->opposite()->facet();
                    // edgeIndex[idx++] = hit->poolId;
                    ids.push_back(hit->poolId);
                    // ids.push_back(hit->opposite_);
                    hit->setVisited();
                    hit->opposite()->setRemoved();
                    // fit2->setRemoved();
                    hit->vertex()->eraseHalfedgeByPointer(hit);
                    hit->opposite()->vertex()->eraseHalfedgeByPointer(hit->opposite());
                    fqueue.push(fit2);
                } else if (hit->opposite()->isAdded() && !hit->opposite()->isVisited()) {
                    ids.push_back(hit->poolId);
                    // ids.push_back(hit->opposite_);
                    hit->opposite()->setVisited();
                    hit->setRemoved();
                    // fit2->setRemoved();
                    hit->vertex()->eraseHalfedgeByPointer(hit);
                    hit->opposite()->vertex()->eraseHalfedgeByPointer(hit->opposite());
                    fqueue.push(fit2);
                }
            }
        }
        if (!ids.empty()) {
            stIndex.push_back(edgeIndexes.size());
            for (int j = 0; j < ids.size(); j++) {
                edgeIndexes.push_back(ids[j]);
            }
            thNumber.push_back(ids.size());
        }
    }

    logt("%d collect halfedge information", start, i_curDecimationId);
    int* dedgeIndexes;
    int* dstIndex;
    int* dthNumber;
    std::vector<int> edgeIndexesCnt(inserted_edgecount, 0);
    CHECK(hipMalloc(&dedgeIndexes, edgeIndexes.size() * sizeof(int)));
    CHECK(hipMalloc(&dstIndex, stIndex.size() * sizeof(int)));
    CHECK(hipMalloc(&dthNumber, thNumber.size() * sizeof(int)));
    CHECK(hipMemcpy(dedgeIndexes, edgeIndexes.data(), edgeIndexes.size() * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dstIndex, stIndex.data(), stIndex.size() * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dthNumber, thNumber.data(), thNumber.size() * sizeof(int), hipMemcpyHostToDevice));
    int vsize = MCGAL::contextPool.vindex;
    int hsize = MCGAL::contextPool.hindex;
    int fsize = MCGAL::contextPool.findex;
    int num = stIndex.size();
    CHECK(hipMemcpy(MCGAL::contextPool.dvpool, MCGAL::contextPool.vpool, vsize * sizeof(MCGAL::Vertex),
                     hipMemcpyHostToDevice));
    CHECK(hipMemcpy(MCGAL::contextPool.dhpool, MCGAL::contextPool.hpool, hsize * sizeof(MCGAL::Halfedge),
                     hipMemcpyHostToDevice));
    CHECK(hipMemcpy(MCGAL::contextPool.dfpool, MCGAL::contextPool.fpool, fsize * sizeof(MCGAL::Facet),
                     hipMemcpyHostToDevice));
    dim3 block(256, 1, 1);
    dim3 grid((num + block.x - 1) / block.x, 1, 1);
    logt("%d cuda memcpy copy", start, i_curDecimationId);
    joinFacetOnCuda<<<grid, block>>>(MCGAL::contextPool.dvpool, MCGAL::contextPool.dhpool, MCGAL::contextPool.dfpool,
                                     dedgeIndexes, dstIndex, dthNumber, num, clockRate);
    hipDeviceSynchronize();
    logt("%d join facet kernel", start, i_curDecimationId);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("ERROR: %s:%d,", __FILE__, __LINE__);
        printf("code:%d,reason:%s\n", error, hipGetErrorString(error));
        exit(1);
    }
    CHECK(hipMemcpy(MCGAL::contextPool.vpool, MCGAL::contextPool.dvpool, vsize * sizeof(MCGAL::Vertex),
                     hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(MCGAL::contextPool.hpool, MCGAL::contextPool.dhpool, hsize * sizeof(MCGAL::Halfedge),
                     hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(MCGAL::contextPool.fpool, MCGAL::contextPool.dfpool, fsize * sizeof(MCGAL::Facet),
                     hipMemcpyDeviceToHost));
    hipFree(dedgeIndexes);
    hipFree(dstIndex);
    hipFree(dthNumber);
    // exit(0);
    return;
}

/**
 * Insert center vertices.
 */
void HiMesh::insertRemovedVertices() {
    // Add the first halfedge to the queue.
    pushHehInit();
    while (!gateQueue.empty()) {
        MCGAL::Halfedge* h = gateQueue.front();
        gateQueue.pop();

        MCGAL::Facet* f = h->facet();

        // If the face is already processed, pick the next halfedge:
        if (f->isProcessed())
            continue;

        // Mark the face as processed.
        f->setProcessedFlag();

        // Add the other halfedges to the queue
        MCGAL::Halfedge* hIt = h;
        do {
            MCGAL::Halfedge* hOpp = hIt->opposite();
            // TODO: wait
            // assert(!hOpp->is_border());
            if (!hOpp->facet()->isProcessed())
                gateQueue.push(hOpp);
            hIt = hIt->next();
        } while (hIt != h);
        assert(!h->isNew());

        if (f->isSplittable()) {
            // Insert the vertex.
            MCGAL::Halfedge* hehNewVertex = create_center_vertex(h);
            hehNewVertex->vertex()->setPoint(f->getRemovedVertexPos());

            // Mark all the created edges as new.
            MCGAL::Vertex* Hvc = hehNewVertex->vertex();
            for (int i = 0; i < Hvc->halfedges_size; i++) {
                MCGAL::Halfedge* hit = Hvc->getHalfedgeByIndex(i);
                hit->setNew();
                hit->opposite()->setNew();
                hit->facet()->setProcessedFlag();
            }
        }
    }
}

/**
 * Remove all the marked edges
 */
void HiMesh::removeInsertedEdges() {
    pushHehInit();
    while (!gateQueue.empty()) {
        MCGAL::Halfedge* h = gateQueue.front();
        gateQueue.pop();

        if (h->isVisited())
            continue;

        if (h->isRemoved()) {
            continue;
        }
        // Mark the face as processed.
        h->setVisited();

        // Add the other halfedges to the queue
        MCGAL::Halfedge* hIt = h;
        do {
            MCGAL::Halfedge* hOpp = hIt->opposite();
            // TODO: wait
            // assert(!hOpp->is_border());
            if (!hOpp->isVisited())
                gateQueue.push(hOpp);
            hIt = hIt->next();
        } while (hIt != h);

        if (hIt->isRemoved()) {
            hIt->setVisited();
            continue;
        }
        if (hIt->isAdded()) {
            join_face(hIt);
            hIt->setVisited();
        }
    }
    return;
}
#include "hip/hip_runtime.h"
//
// Created by DELL on 2023/11/9.
//
#include "himesh.cuh"
#include "math.h"
#include "util.h"

void HiMesh::encode(int lod) {
    b_jobCompleted = false;
    int hid = 0;
    // set id for facet and halfedge
    for (int i = 0; i < faces.size(); i++) {
        faces[i]->setFid(i);
        for (int j = 0; j < faces[i]->halfedge_size; j++) {
            faces[i]->getHalfedgeByIndex(j)->setHid(hid++);
        }
    }

    while (!b_jobCompleted) {
        startNextCompresssionOp();
    }
}

void HiMesh::startNextCompresssionOp() {
    // 1. reset the stats
    for (MCGAL::Vertex* vit : vertices) {
        vit->resetState();
    }
    for (auto fit = faces.begin(); fit != faces.end();) {
        if ((*fit)->isRemoved()) {
            fit = faces.erase(fit);
        } else {
            (*fit)->resetState();
            for (int i = 0; i < (*fit)->halfedge_size; i++) {
                (*fit)->getHalfedgeByIndex(i)->resetState();
            }
            fit++;
        }
    }
    i_nbRemovedVertices = 0;  // Reset the number of removed vertices.
    while (!gateQueue.empty()) {
        gateQueue.pop();
    }

    // 2. do one round of decimation
    if (i_curDecimationId < 10) {
        MCGAL::Facet* fit = faces[faces.size() / 2];
        fit->getHalfedgeByIndex(0)->setInQueue();
        gateQueue.push(fit->getHalfedgeByIndex(0));
    }
    // bfs all the facet
    while (!gateQueue.empty()) {
        MCGAL::Halfedge* h = gateQueue.front();
        gateQueue.pop();
        // TODO: wait
        // assert(!h->is_border());
        MCGAL::Facet* f = h->facet();
        // if the face is already processed, pick the next halfedge:
        if (f->isConquered()) {
            h->removeFromQueue();
            continue;
        }
        // the face is not processed. Count the number of non conquered vertices that can be split
        bool hasRemovable = false;
        MCGAL::Halfedge* unconqueredVertexHE;

        for (MCGAL::Halfedge* hh = h->next(); hh != h; hh = hh->next()) {
            if (isRemovable(hh->end_vertex())) {
                hasRemovable = true;
                unconqueredVertexHE = hh;
                break;
            }
        }

        // if all face vertices are conquered, then the current face is a null patch:
        if (!hasRemovable) {
            f->setUnsplittable();
            // and add the outer halfedges to the queue. Also mark the vertices of the face conquered
            MCGAL::Halfedge* hh = h;
            do {
                hh->vertex()->setConquered();
                MCGAL::Halfedge* hOpp = hh->opposite();
                // TODO: wait
                // assert(!hOpp->is_border());
                if (!hOpp->facet()->isConquered()) {
                    gateQueue.push(hOpp);
                    hOpp->setInQueue();
                }
            } while ((hh = hh->next()) != h);
            h->removeFromQueue();
        } else {
            // in that case, cornerCut that vertex.
            h->removeFromQueue();
            vertexCut(unconqueredVertexHE);
        }
    }
    // 3. do the encoding job
    if (i_nbRemovedVertices == 0) {
        b_jobCompleted = true;
        i_nbDecimations = i_curDecimationId--;
        // Write the compressed data to the buffer.
        writeBaseMesh();
        int i_deci = i_curDecimationId;
        assert(i_deci >= 0);
        while (i_deci >= 0) {
            // 先写facet_number halfedge_number
            // halfedge_number
            int before = *dataOffset;
            encodeRemovedVertices(i_deci);
            int after = *dataOffset;
            int cur_totalOffset = facetOffset[i_deci].size() * sizeof(int) + sizeof(int);
            // 先拷贝
            memcpy(p_data + before + cur_totalOffset, p_data + before, after - before);
            *dataOffset = before;
            // facet_number
            writeInt(facetOffset[i_deci].size());
            encodeFacetOffset(i_deci);
            i_deci--;
            *dataOffset += after - before;
            writeInt(encodeEdgeId[i_deci].size());
            encodeInsertedEdges(i_deci);
        }
    } else {
        // 3dpro: compute and encode the Hausdorff distance for all the facets in this LOD
        // computeHausdorfDistance();
        // HausdorffCodingStep();
        // RemovedVertexCodingStep();
        InsertedEdgeCodingStep();
        // finish this round of decimation and start the next
        i_curDecimationId++;  // Increment the current decimation operation id.
    }
}

MCGAL::Halfedge* HiMesh::vertexCut(MCGAL::Halfedge* startH) {
    MCGAL::Vertex* v = startH->end_vertex();

    // make sure that the center vertex can be removed
    assert(!v->isConquered());
    assert(v->vertex_degree() > 2);

    MCGAL::Halfedge* h = startH->opposite();
    MCGAL::Halfedge* end(h);
    int removed = 0;
    int count = 0;
    do {
        // TODO: wait
        // assert(!h->is_border());
        MCGAL::Facet* f = h->facet();
        assert(!f->isConquered() && !f->isRemoved());  // we cannot cut again an already cut face, or a NULL patch
        // if the face is not a triangle, cut the corner to make it a triangle
        if (f->facet_degree() > 3) {
            // loop around the face to find the appropriate other halfedge
            MCGAL::Halfedge* hSplit(h->next());
            for (; hSplit->next()->next() != h; hSplit = hSplit->next())
                ;
            MCGAL::Halfedge* hCorner = split_facet(h, hSplit);
            // mark the new halfedges as added
            hCorner->setAdded();
            hCorner->opposite()->setAdded();
            // the corner one inherit the original facet
            // while the fRest is a newly generated facet
        }
        // mark the vertex as conquered
        h->end_vertex()->setConquered();
        // h->end_vertex->setConquered();
        removed++;
        count++;
    } while ((h = h->opposite()->next()) != end);

    // copy the position of the center vertex:
    MCGAL::Point vPos = startH->end_vertex()->point();
    // 记录所有需要被移除的信息
    std::vector<int> encode;
    encode.reserve(3 * count);
    // 首先记录起始的halfedgeId
    encode.push_back(find_prev(startH)->hid);
    pre_erase_center_vertex(startH, encode);
    encodeQueue[i_curDecimationId].push_back(encode);
    // remove the center vertex
    MCGAL::Halfedge* hNewFace = erase_center_vertex(startH);
    MCGAL::Facet* added_face = hNewFace->facet();

    // now mark the new face as having a removed vertex
    added_face->setSplittable();
    // keep the removed vertex position.
    added_face->setRemovedVertexPos(vPos);
    geometrySym[i_curDecimationId].push_back(vPos);
    // scan the outside halfedges of the new face and add them to
    // the queue if the state of its face is unknown. Also mark it as in_queue
    h = hNewFace;
    do {
        MCGAL::Halfedge* hOpp = h->opposite();
        // TODO: wait
        // assert(!hOpp->is_border());
        if (!hOpp->facet()->isConquered()) {
            gateQueue.push(hOpp);
            hOpp->setInQueue();
        }
    } while ((h = h->next()) != hNewFace);
    // Increment the number of removed vertices.
    i_nbRemovedVertices++;
    removedPoints.push_back(vPos);
    return hNewFace;
}

void HiMesh::InsertedEdgeCodingStep() {
    encodeEdgeId.push_back(std::deque<int>());
    pushHehInit();
    while (!gateQueue.empty()) {
        MCGAL::Halfedge* h = gateQueue.front();
        gateQueue.pop();
        if (h->isProcessed()) {
            continue;
        }
        // Mark the halfedge as processed.
        h->setProcessed();
        h->opposite()->setProcessed();

        // Add the other halfedges to the queue
        MCGAL::Halfedge* hIt = h->next();
        while (hIt->opposite() != h) {
            if (!hIt->isProcessed())
                gateQueue.push(hIt);
            hIt = hIt->opposite()->next();
        }

        // Don't write a symbol if the two faces of an edgde are unsplitable.
        // this can help to save some space, since it is guaranteed that the edge is not inserted
        bool b_toCode = h->facet()->isUnsplittable() && h->opposite()->facet()->isUnsplittable() ? false : true;

        // Determine the edge symbol.
        unsigned sym;
        if (!h->isOriginal())
            encodeEdgeId[i_curDecimationId].push_back(h->hid);
    }
}

void HiMesh::writeBaseMesh() {
    unsigned i_nbVerticesBaseMesh = size_of_vertices();
    unsigned i_nbFacesBaseMesh = size_of_facets();
    // Write the number of level of decimations.
    writeInt16(i_nbDecimations);

    // Write the number of vertices and faces on 16 bits.
    writeInt(i_nbVerticesBaseMesh);
    writeInt(i_nbFacesBaseMesh);
    size_t id = 0;
    for (unsigned j = 0; j < 2; ++j) {
        MCGAL::Point p = vh_departureConquest[j]->point();
        writePoint(p);
        vh_departureConquest[j]->setId(id++);
    }
    // Write the other vertices.
    for (MCGAL::Vertex* vit : vertices) {
        if (vit == vh_departureConquest[0] || vit == vh_departureConquest[1])
            continue;
        MCGAL::Point point = vit->point();
        writePoint(point);
        // Set an id to the vertex.
        vit->setId(id++);
    }
    // Write the base mesh face vertex indices.
    for (MCGAL::Facet* fit : faces) {
        unsigned i_faceDegree = fit->facet_degree();
        writeInt(fit->fid);
        writeInt(i_faceDegree);
        MCGAL::Halfedge* st = fit->getHalfedgeByIndex(0);
        MCGAL::Halfedge* ed = st;
        do {
            writeInt(st->hid);
            st = st->next();
        } while (st != ed);
        st = fit->getHalfedgeByIndex(0);
        do {
            writeInt(st->vertex()->getId());
            st = st->next();
        } while (st != ed);
    }
}

/**
 * Encode an inserted edge list.
 */
void HiMesh::encodeInsertedEdges(unsigned i_operationId) {
    std::deque<int>& edgeId = encodeEdgeId[i_operationId];
    assert(edgeId.size() > 0);

    unsigned i_len = edgeId.size();
    for (unsigned i = 0; i < i_len; ++i) {
        writeInt(edgeId[i]);
    }
}

/**
 * Encode the geometry and the connectivity of a removed vertex list.
 */
void HiMesh::encodeRemovedVertices(unsigned i_operationId) {
    std::deque<std::vector<int>>& encodeIds = encodeQueue[i_operationId];
    std::deque<MCGAL::Point>& geomSym = geometrySym[i_operationId];
    std::deque<int>& faceOffset = facetOffset[i_operationId];
    unsigned i_lenGeom = geomSym.size();
    unsigned i_lenConn = encodeIds.size();
    assert(i_lenGeom > 0);
    assert(i_lenConn > 0);
    unsigned k = 0;
    for (unsigned i = 0; i < i_lenConn; ++i) {
        faceOffset.push_back(*dataOffset);
        // Encode the connectivity.
        for (int j = 0; j < encodeIds[i].size(); j++) {
            writeInt(encodeIds[i][j]);
        }
        writePoint(geomSym[k]);
        k++;
    }
}

/**
 * Encode the geometry and the connectivity of a removed vertex list.
 */
void HiMesh::encodeFacetOffset(unsigned i_operationId) {
    std::deque<int>& faceOffset = facetOffset[i_operationId];
    unsigned i_lenConn = faceOffset.size();
    assert(i_lenConn > 0);
    for (unsigned i = 0; i < i_lenConn; ++i) {
        // Encode the connectivity.
        unsigned sym = faceOffset[i];
        writeInt(sym);
    }
}
#include "hip/hip_runtime.h"
#include "../PPMC/BATCH_PPMC/DecompressTool.cuh"
#include <iostream>
using namespace std;

__global__ void kernel() {
    printf("hello world\n");
}

void compress(int argc, char** argv) {
    // const char* strings[] = {
    //     "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer1", "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer2",
    //     "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer3", "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer4",
    //     "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer5", "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer6",
    //     "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer7", "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer8",
    //     "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer9", "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer10",
    //     "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer11", "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer12",
    //     "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer13", "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer14",
    //     "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer15", "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer16",
    //     "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer17", "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer18",
    //     "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer19", "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer20"};
    // const char* strings[] = {
    //     "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer1", "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer2",
    //     "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer3", "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer4",
    //     "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer5", "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer6",
    //     "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer7", "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer8",
    //     "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer9", "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer10"};
    const char* strings[] = {
        "/home/koi/mastercode/Mini_CGAL/buffers/avoidCompetitionBuffer1", "/home/koi/mastercode/Mini_CGAL/buffers/avoidCompetitionBuffer2",
        "/home/koi/mastercode/Mini_CGAL/buffers/avoidCompetitionBuffer3", "/home/koi/mastercode/Mini_CGAL/buffers/avoidCompetitionBuffer4",
        "/home/koi/mastercode/Mini_CGAL/buffers/avoidCompetitionBuffer5", "/home/koi/mastercode/Mini_CGAL/buffers/avoidCompetitionBuffer6",
        "/home/koi/mastercode/Mini_CGAL/buffers/avoidCompetitionBuffer7", "/home/koi/mastercode/Mini_CGAL/buffers/avoidCompetitionBuffer8",
        "/home/koi/mastercode/Mini_CGAL/buffers/avoidCompetitionBuffer9", "/home/koi/mastercode/Mini_CGAL/buffers/avoidCompetitionBuffer10"};
    // const char* strings[] = {
    //     "/home/koi/mastercode/Mini_CGAL/buffers/avoidCompetitionBuffer11"};
    // const char* strings[] = {
    //     "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer1", "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer2",
    //     "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer3"};
    //const char* strings[] = {
    //     "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer1", "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer2",
    //     "/home/koi/mastercode/Mini_CGAL/buffers/newbuffer3"};
    char** paths = new char*[sizeof(strings) / sizeof(strings[0])];
    for (size_t i = 0; i < sizeof(strings) / sizeof(strings[0]); ++i) {
        paths[i] = new char[strlen(strings[i]) + 1];
        strcpy(paths[i], strings[i]);
    }
    DeCompressTool* deCompressTool = new DeCompressTool(paths, 10, true);
    int lod = 100;
    char path[256];
    sprintf(path, "%s", "./gisdata/compressed_0_mesh_%d_mesh.off");
    // deCompressTool->dumpto(path);
    struct timeval start = get_cur_time();
    for (uint i = 10; i <= lod; i += 10) {
        deCompressTool->decode(i);
        logt("decode to %d", start, i);
        sprintf(path, "./gisdata/compressed_%d%s", i, "_mesh_%d_mesh.off");
        // std::cout << path << std::endl;
        // printf("%s", path);
        // deCompressTool->dumpto(path);
    }
    delete deCompressTool;
}

int main(int argc, char** argv) {
    // kernel<<<1, 1>>>();
    compress(argc, argv);
    return 0;
}